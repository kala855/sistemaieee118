#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "utilities/utilities.h"
#include <time.h>
#include <math.h>
extern void dgesv_( int* n, int* nrhs, double* a, int* lda, int* ipiv,\
        double* b, int* ldb, int* info );

extern void dgemv_(char *trans, int *m, int *n, double *alpha, double *a,\
        int *lda, double *x, int *incx, double *beta, double *y, int *incy);


int main(){
    int res,i,j;
    int widthCargas = 3;
    int widthGen = 3;
    int widthLineas = 6;
    int heightCargas = 83;
    int heightGen = 15;
    int heightLineas = 186, NumP;
    char *fileNameLineas = "../../inputs/lineas";
    char *fileNameCargas = "../../inputs/cargas";
    char *fileNameGen = "../../inputs/gen";
    char *fileNameIMax = "../../inputs/imax";
    char *fileNameNW = "../../inputs/NW";
    int numDataImax = 186, NumW = 3;
    double *Imax, *A, *ZpReal, *ZpImag, *NW, *Vn, *An;
    structData *data;
    Mont *mont;
    Imax = malloc(numDataImax*sizeof(double));
    data = (structData*)malloc(sizeof(structData));
    mont = (Mont*)malloc(sizeof(Mont));
    ///////// Los datos son cargados desde los archivos de texto //////////////////////////
    //////// Lineas, generadores y cargas se llevan a la estructura data//////////////////
    res = loadDataFromFile(fileNameLineas,fileNameCargas,fileNameGen, data);
    data->Sbase = 100;
    data->iteraciones = 40;
    data->fnom = 60.0;
    data->maxIter = 100;
    data->numN = maxLineas(data,widthLineas,heightLineas);
    data->numL = heightLineas;
    data->numG = heightGen;
    data->numC = heightCargas;
    Vn = (double*)malloc(data->numN*sizeof(double));
    An = (double*)malloc(data->numN*sizeof(double));
    ones(data->numN,Vn);
    //////////////////////////////////////////////////////////////////////////////////////
    A = malloc((int)(data->numN)*(int)(data->numL)*sizeof(double));
    NW = malloc(3*4*sizeof(double));
    //////////////////////////////////////////////////////////////////////////////////////

    //////////////////////////////////////////////////////////////////////////////////////
    res = loadCorrientesMax(fileNameIMax, Imax);
    res = loadNW(fileNameNW, NW);

/*    for (i = 0; i < 3; i++) {
       for (j = 0; j < 4; j++) {
           printf("%lf ", NW[i*4+j]);
       }
       printf("\n");
    }*/

    //////////////////////////////////////////////////////////////////////////////////////
    calcularMatrizA(data,widthLineas,A);
    printMatrixToFile(A,data->numL, data->numN, "A");
    //////////////////////////////////////////////////////////////////////////////////////
    ZpReal = malloc(heightLineas*sizeof(double));
    ZpImag = malloc(heightLineas*sizeof(double));
    calcularZp(data,heightLineas,widthLineas,ZpReal,ZpImag);


    //dgemv_ configuration parameters
    int mi = 186, n = 118, lda = 186, ldan = 118, incx = 1, incy = 1;
    double alpha = 1, beta = 0, *Vrama, *Irama, *VnReal, *VnImag, *ybusReal;
    double *ybusImag, *InReal, *InImag, *InImagTmp, *IlineaReal, *IlineaImag, *sobrecarga, *Vmedia;
    double *Vdesv, *Probmin, *Probmax, *Probsobrecarga;

    int ni = 1000, m;
    int k;
    char nameVrama[10], nameIrama[10], nameZpReal[10], nameVnReal[10], nameVnImag[10];
    char nameInReal[10], nameInImag[10], nameSobrecarga[10];
    double r, lambda = 0.0, kk, N, Pmax, Vw, Pw;
    NumW = 3;
    double Vmin = 4, Vnom = 12, Vmax = 25;
    srand((unsigned)time(NULL));
    ybusReal = malloc(data->numN*data->numN*sizeof(double));
    ybusImag = malloc(data->numN*data->numN*sizeof(double));
    InReal = malloc(data->numN*sizeof(double));
    InImag = malloc(data->numN*sizeof(double));
    InImagTmp = malloc(data->numN*sizeof(double));
    Vrama = malloc(data->numL*sizeof(double));
    Irama = malloc(data->numL*sizeof(double));
    IlineaReal = malloc(data->numL*sizeof(double));
    IlineaImag = malloc(data->numL*sizeof(double));
    sobrecarga = malloc(data->numL*sizeof(double));
    VnReal = malloc(data->numN*sizeof(double));
    VnImag = malloc(data->numN*sizeof(double));
    mont->sum = malloc(data->numN*sizeof(double));
    mont->sumcuad = malloc(data->numN*sizeof(double));
    mont->lv = malloc(data->numN*sizeof(double));
    mont->hv = malloc(data->numN*sizeof(double));
    mont->sob = malloc(data->numL*sizeof(double));
    Vmedia = malloc(data->numN*sizeof(double));
    Vdesv = malloc(data->numN*sizeof(double));
    Probmin = malloc(data->numN*sizeof(double));
    Probmax = malloc(data->numN*sizeof(double));
    Probsobrecarga = malloc(data->numL*sizeof(double));

    //mont Inicializacion
    zeros(data->numN, mont->sum);
    zeros(data->numN, mont->sumcuad);
    zeros(data->numN, mont->lv);
    zeros(data->numN, mont->hv);
    zeros(data->numL, mont->sob);

    for (k = 0; k < ni; k++) {
        for (m = 0; m < NumW; m++) {
            r = ((double)rand()/(double)RAND_MAX);
            lambda = NW[m*NumW+2];
            kk = NW[m*NumW+3];
            Pmax = NW[m*NumW+1];
            N = NW[m*NumW+0] - 1.0;
            Vw = lambda*pow((-log(1-r)),(1/kk));
            if ((Vw<Vmin)||(Vw>Vmax))
                Pw = 0.0;
            if ((Vmin<Vw)&&(Vw<Vnom))
                Pw = Pmax*pow((Vw/Vnom),3);
            if ((Vw>Vnom)&&(Vw<Vmax))
                Pw = Pmax;

            data->gen[(int)(N)*widthGen+1] = Pw;
            //printf("%lf \n",data->gen[(int)(N)*widthGen+1]);
        }
        // Llamar versión paralela
        res = newtonRaphson(data, Vn, An, ybusReal, ybusImag);
        calculoVn(Vn,An, data->numN, VnReal, VnImag);
        //cálculo de InReal --> Cálculo a Realizar en GPU
        dgemv_("N",&n, &n, &alpha, ybusReal, &ldan, VnReal, &incx, &beta, InReal, &incy);
        dgemv_("N",&n, &n, &alpha, ybusImag, &ldan, VnImag, &incx, &beta, InImag, &incy);
        res = subVectors(InReal, InImag, data->numN);

        //cálculo de InImag --> Cálculo a Realizar en GPU
        dgemv_("N",&n, &n, &alpha, ybusReal, &ldan, VnImag, &incx, &beta, InImag, &incy);
        dgemv_("N",&n, &n, &alpha, ybusImag, &ldan, VnReal, &incx, &beta, InImagTmp, &incy);
        res = addVectors(InImag,InImagTmp, data->numN);

        //cálculo de ilinea --> Cálculo a Realizar en GPU
        dgemv_("N",&mi, &n, &alpha, A, &lda, InReal, &incx, &beta, IlineaReal, &incy);
        dgemv_("N",&mi, &n, &alpha, A, &lda, InImag, &incx, &beta, IlineaImag, &incy);

        // Cálculo a Realizar en GPU
        calculoSobrecarga(IlineaReal, IlineaImag, sobrecarga, Imax, data->numL);
        calculoMontSobrecarga(data->numL, sobrecarga, mont);

        //Cálculo a Realizar en GPU
        calculoCorrientesRama(data->numN,Vn,mont);

  //      sprintf(nameSobrecarga, "sobrecarga%d",k);
        //sprintf(nameVnReal, "VnReal%d",k);
        //sprintf(nameVnImag, "VnImag%d",k);
        //sprintf(nameInReal, "InReal%d",k);
        //sprintf(nameInImag, "InImag%d",k);
        //sprintf(nameVrama, "Vrama%d",k);
        //sprintf(nameIrama, "Irama%d",k);
        //sprintf(nameZpReal, "Zp%d",k);
//        printDataToFileVec(nameSobrecarga,data->numL,mont->sob);
       // printDataToFileVec(name,data->numN,Vn);
        //printDataToFileVec(nameVnReal,data->numN,VnReal);
        //printDataToFileVec(nameVnReal,data->numN,VnReal);
        //printDataToFileVec(nameInReal,data->numL, IlineaReal);
        //printDataToFileVec(nameInImag,data->numL, IlineaImag);
        //dgemv_("N",&mi, &n, &alpha, A, &lda, Vn, &incx, &beta, Vrama, &incy);
        //printDataToFileVec(nameVrama,186,Vrama);
        //calculoIrama(Vrama,ZpReal,ZpImag,heightLineas,Irama);
        //printDataToFileVec(nameIrama,186,Irama);
        //printDataToFileVec(nameZpReal,186,ZpReal);
    }
    /////////////////////////////////////////////////////////////////////////////////////

    calculosFinales(data->numN, ni, mont,Vmedia,Vdesv,Probmin,Probmax);
    calculoProbSobrecarga(data->numL,ni,mont, Probsobrecarga);
    printDataToFileVec("vmedia",data->numN,Vmedia);
    printDataToFileVec("vdesv", data->numN,Vdesv);
    printDataToFileVec("probmin", data->numN,Probmin);
    printDataToFileVec("probmax", data->numN,Probmax);
    printDataToFileVec("probsobrecarga", data->numL,Probsobrecarga);

    free(Imax);
    free(VnReal);free(VnImag);
    free(ybusReal);free(ybusImag);
    free(InReal);free(InImag);free(InImagTmp);
    free(IlineaReal);free(IlineaImag);
    free(mont->sob);free(mont->lv);free(mont->sum);free(mont->sumcuad);free(mont->hv);free(mont);
    free(Vmedia);free(Vdesv);free(Probmin);free(Probmax);free(Probsobrecarga);
    free(sobrecarga);
    free(Irama);
    free(Vrama);
    free(data);
    free(Vn);
    return res;
}
