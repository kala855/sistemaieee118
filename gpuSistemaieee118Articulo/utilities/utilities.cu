#include "hip/hip_runtime.h"
#include <stdlib.h>
#include "utilities.h"
#include <cusolverUtilities.cuh>
#include <string.h>
#include <math.h>
extern void dgesv_( int* n, int* nrhs, double* a, int* lda, int* ipiv,\
        double* b, int* ldb, int* info );



int calcCargLineas(structData *data, double *An, double *Vn, double *Ism){
    int k, N1, N2;
    double akm;
    int widthLineas = 6;
    for (k = 0; k < data->numL; k++) {
        N1 = data->lineas[k*widthLineas+0] - 1;
        N2 = data->lineas[k*widthLineas+1] - 1;
        akm = An[N1] - An[N2];
        Ism[k] = sqrt(Vn[N1]*Vn[N1] + Vn[N2]*Vn[N2] - 2*Vn[N1]*Vn[N2]*cos(akm));
        Ism[k] = Ism[k]/sqrt(data->lineas[k*widthLineas+2]*data->lineas[k*widthLineas+2] + \
                data->lineas[k*widthLineas+3]*data->lineas[k*widthLineas+3])\
                 /1.6;//data->lineas[k*widthLineas+6];

    }
    return 0;
}

double maxAbs(int NumPQ, double *dPdQ){
    int i;
    double max = 0.0;
    for (i = 0; i < NumPQ; i++){
        if(fabs(dPdQ[i]) > max)
            max = fabs(dPdQ[i]);
    }
    return max;
}

int transposeJacR(double *JacR,int NumPQ, double *JacRt){
    int i, j;
    for ( i = 0; i < NumPQ; i++) {
        for (j = 0; j < NumPQ; j++) {
            JacRt[i*NumPQ+j] = JacR[j*NumPQ+i];
        }
    }
    return 0;

}

int createdPdQ(double *dP, double *dQ, int NumP, int NumQ, double *dPdQ){
    memcpy(dPdQ,dP,sizeof(double)*NumP);
    memcpy(dPdQ+NumP,dQ,sizeof(double)*NumQ);
    return 0;
}

int createJacR(int *NNP, int *NNQ, int NumQ, int NumP, int numN,double *Jpp, \
        double *Jpq, double *Jqp, double *Jqq, double *JacR){
   // printf("%d %d\n", NumQ, NumP);
    int size = NumP+NumQ;
    int i,j,k,l;
    for (i = 0; i < NumP; i++) {
        for (j = 0; j < NumP; j++) {
            JacR[i*size+j] = Jpp[(NNP[i]-1)*numN+(NNP[j]-1)];
        }
    }

    k = 0;
    for (i = 0; i < NumP; i++) {
        l = NumP;
        for (j = 0; j < NumQ; j++) {
            JacR[k*size+l] = Jpq[(NNP[i]-1)*numN+(NNQ[j]-1)];
            l++;
        }
        k++;
    }

    k = NumP;
    for (i = 0; i < NumQ; i++) {
        l = 0;
        for (j = 0; j < NumP; j++) {
            JacR[k*size+l] = Jqp[(NNQ[i]-1)*numN+(NNP[j]-1)];
            l++;
        }
        k++;
    }

    k = NumP;
    for (i = 0; i < NumQ; i++) {
        l = NumP;
        for (j = 0; j < NumQ; j++) {
            JacR[k*size+l] = Jqq[(NNQ[i]-1)*numN+(NNQ[j]-1)];
            l++;
        }
        k++;
    }



    return 0;
}

int calcularJacobiano(structData *data, double *ybusReal, double *ybusImag, double *Vn, double *An, \
        double *Jpp, double *Jpq, double *Jqp, double *Jqq, double *Pn, double *Qn){

    zeros(data->numN,Pn);
    zeros(data->numN,Qn);
    double akm, *H,*N,*J,*L;
    int k,m,widthLineas=data->numN;
    for (k = 0; k < data->numN; k++) {
        for (m = 0; m < data->numN; m++) {
            akm = An[k] - An[m];
            Pn[k] = Pn[k] + Vn[m]*(ybusReal[k*widthLineas+m]*cos(akm) + \
                    ybusImag[k*widthLineas+m]*sin(akm));
            Qn[k] = Qn[k] - Vn[m]*(ybusImag[k*widthLineas+m]*cos(akm) - \
                    ybusReal[k*widthLineas+m]*sin(akm));
        }
        Pn[k] = Pn[k]*Vn[k];
        Qn[k] = Qn[k]*Vn[k];
    }

    H = (double*)malloc(data->numN*data->numN*sizeof(double));
    N = (double*)malloc(data->numN*data->numN*sizeof(double));
    J = (double*)malloc(data->numN*data->numN*sizeof(double));
    L = (double*)malloc(data->numN*data->numN*sizeof(double));

    zeros(data->numN*data->numN,H);
    zeros(data->numN*data->numN,N);
    zeros(data->numN*data->numN,J);
    zeros(data->numN*data->numN,L);

    for (k = 0; k < data->numN; k++) {
        for (m = 0; m < data->numN; m++) {
            if(k==m){
                H[k*widthLineas+k] = -ybusImag[k*widthLineas+k]*Vn[k]*Vn[k]-Qn[k];
                N[k*widthLineas+k] = ybusReal[k*widthLineas+k]*Vn[k] + Pn[k]/Vn[k];
                J[k*widthLineas+k] = -ybusReal[k*widthLineas+k]*Vn[k]*Vn[k] + Pn[k];
                L[k*widthLineas+k] = -ybusImag[k*widthLineas+k]*Vn[k] + Qn[k]/Vn[k];
            }else{
                akm = An[k] - An[m];
                H[k*widthLineas+m] = Vn[k]*Vn[m]*(ybusReal[k*widthLineas+m]*sin(akm)- \
                        ybusImag[k*widthLineas+m]*cos(akm)) ;
                N[k*widthLineas+m] = Vn[k]*(ybusReal[k*widthLineas+m]*cos(akm)+ybusImag[k*widthLineas+m]*sin(akm));
                J[k*widthLineas+m] = -N[k*widthLineas+m]*Vn[m];
                L[k*widthLineas+m] = H[k*widthLineas+m]/Vn[m];
            }
        }
    }

    memcpy(Jpp,H,data->numN*data->numN*sizeof(double));
    memcpy(Jpq,N,data->numN*data->numN*sizeof(double));
    memcpy(Jqp,J,data->numN*data->numN*sizeof(double));
    memcpy(Jqq,L,data->numN*data->numN*sizeof(double));

    free(H);free(N);free(J);free(L);
    return 0;

}

int setdiff(int *vector1, double *vector2, int size1, int size2, int *c){
    int i, j, accum=0, k=0;
    for (i = 0; i < size1; i++) {
        for (j = 0; j < size2; j++) {
            if(vector1[i]!=(int)vector2[j*3])
                accum++;
            else
                break;
        }
        if(accum == size2){
            c[k] = vector1[i];
            k++;
        }
        accum = 0;
    }
    return k;
}

int genVector(int *NNP, int initNumber,int finalNumber){
    int i = initNumber,j=0;
    for (i = initNumber; i <= finalNumber; i++) {
        NNP[j] = i;
        j++;
    }
    return 0;
}

int calcularYbus(structData *data, double *ybusReal, double *ybusImag){
    zeros(data->numN*data->numN, ybusReal);
    zeros(data->numN*data->numN, ybusImag);
    int widthLineas = 6;
    double N1, N2, ym;
    double YYReal, YYImag, tap;
    int k,t;
    for (k = 0; k < data->numL; k++) {
        N1 = data->lineas[k*widthLineas+0]-1;
        N2 = data->lineas[k*widthLineas+1]-1;
        ym = data->lineas[k*widthLineas+2]*data->lineas[k*widthLineas+2]+\
             data->lineas[k*widthLineas+3]*data->lineas[k*widthLineas+3];
        YYReal = data->lineas[k*widthLineas+2]/ym;
        YYImag = -data->lineas[k*widthLineas+3]/ym;

        tap = 1/data->lineas[k*widthLineas+5];

        ybusReal[(int) (N1*data->numN + N1)] = ybusReal[(int) (N1*data->numN + N1)]\
                                               + tap*tap*YYReal;
        ybusReal[(int) (N2*data->numN + N2)] = ybusReal[(int) (N2*data->numN + N2)] \
                                               + YYReal;
        ybusReal[(int) (N1*data->numN + N2)] = ybusReal[(int) (N1*data->numN + N2)] \
                                               - tap*YYReal;
        ybusReal[(int) (N2*data->numN + N1)] = ybusReal[(int) (N2*data->numN + N1)] \
                                               - tap*YYReal;

        ybusImag[(int) (N1*data->numN + N1)] = ybusImag[(int) (N1*data->numN + N1)] \
                                               + tap*tap*YYImag + data->lineas[k*widthLineas+4];
        ybusImag[(int) (N2*data->numN + N2)] = ybusImag[(int) (N2*data->numN + N2)] \
                                               + YYImag + data->lineas[k*widthLineas+4];
        ybusImag[(int) (N1*data->numN + N2)] = ybusImag[(int) (N1*data->numN + N2)] \
                                               - tap*YYImag;
        ybusImag[(int) (N2*data->numN + N1)] = ybusImag[(int) (N2*data->numN + N1)] \
                                               - tap*YYImag;
    }

    return 0;


}

int zeros(int size,double *An){
    int i;
    for (i = 0; i < size; i++) {
        An[i] = 0.0;
    }
    return 0;
}

int ones(int size,double *Vn){
    int i;
    for (i = 0; i < size; i++) {
        Vn[i] = 1.0;
    }
    return 0;
}

double maxLineas(structData *data, int widthLineas,int heightLineas){
    int i;
    double mayorLineas1 = 0, mayorLineas2;
    for (i = 0; i < heightLineas; i++) {
        if(mayorLineas1 < data->lineas[i*widthLineas]){
            mayorLineas1 = data->lineas[i*widthLineas];
        }
        if(mayorLineas2 < data->lineas[i*widthLineas+1]){
            mayorLineas2 = data->lineas[i*widthLineas+1];
        }
    }

    if(mayorLineas2>=mayorLineas1)
        return mayorLineas2;
    else
        return mayorLineas1;

}

int printData(structData *data, int widthLineas, int heightLineas, int widthCargas, int heightCargas, int widthGen, int heightGen){
    int i;
    printf("Datos Líneas\n");
    for (i = 0; i < heightLineas; i++) {
        printf("%.5lf,%.5lf,%.5lf,%.5lf,%.5lf\n",data->lineas[i*widthLineas],data->lineas[i*widthLineas+1],data->lineas[i*widthLineas+2],data->lineas[i*widthLineas+3],data->lineas[i*widthLineas+4]);
    }

    printf("Datos Cargas\n");
    for (i = 0; i < heightCargas; i++) {
        printf("%.5lf,%.5lf,%.5lf\n",data->cargas[i*widthCargas],data->cargas[i*widthCargas+1],data->cargas[i*widthCargas+2]);
    }

    printf("Datos Gen\n");
    for (i = 0; i < heightGen; i++) {
        printf("%.5lf,%.5lf,%.5lf\n",data->gen[i*widthGen],data->gen[i*widthGen+1],data->gen[i*widthGen+2]);
    }

    return 0;
}

int loadCorrientesMax(char *fileNameIMax, double *Imax){
    FILE *datosImax;
    int numDataImax, i;
    numDataImax = 186;
    datosImax = fopen(fileNameIMax,"r");
    if (datosImax == NULL){
        printf("Archivo de Imax inexistente %s verifique \n",fileNameIMax);
        exit(1);
    }

    for(i=0;i<numDataImax;i++){
        fscanf(datosImax,"%lf\n",&Imax[i]);
    }

    fclose(datosImax);
    return 0;
}



int loadDataFromFile(char *filenameLineas, char *filenameCargas, char *filenameGen, structData *data){
    FILE *datosLineas,*datosGen,*datosCargas;
    int i, j, widthLineas,heightLineas,widthGen, heightGen,widthCargas, heightCargas;
    widthLineas = 6;
    heightLineas = 186;
    widthGen = 3;
    heightGen = 15;
    widthCargas = 3;
    heightCargas = 83;

    datosGen = fopen(filenameGen,"r");
    datosLineas = fopen(filenameLineas,"r");
    datosCargas = fopen(filenameCargas,"r");

    if (datosGen == NULL){
        printf("Archivo de Gen inexistente %s verifique \n", filenameGen);
        exit(1);
    }

    if (datosLineas == NULL){
        printf("Archivo de Lineas inexistente %s verifique \n", filenameLineas);
        exit(1);
    }

    if(datosCargas == NULL){
        printf("Archivo de Cargas inexistente %s verifique \n", filenameCargas);
        exit(1);
    }

    data->lineas = malloc(widthLineas*heightLineas*sizeof(double));
    data->cargas = malloc(widthCargas*heightCargas*sizeof(double));
    data->gen = malloc(widthGen*heightGen*sizeof(double));

    if (data->lineas == NULL){
        printf("Imposible asignar memoria a lineas\n");
        exit(1);
    }
    if (data->gen == NULL){
        printf("Imposible asignar memoria a gen\n");
        exit(1);
    }

    if (data->cargas == NULL){
        printf("Imposible asignar memoria a cargas\n");
        exit(1);
    }
    for (i = 0; i < heightLineas; i++) {//Se adiciona el tap constante en 1.0 para este caso
        fscanf(datosLineas, "%lf,%lf,%lf,%lf,%lf\n",&data->lineas[i*widthLineas],\
                &data->lineas[i*widthLineas+1],&data->lineas[i*widthLineas+2],\
                &data->lineas[i*widthLineas+3],&data->lineas[i*widthLineas+4]);
        if(data->lineas[i*widthLineas+4]==0)
            data->lineas[i*widthLineas+4] = 1.0;

        data->lineas[i*widthLineas+5] = 1.0;
    }

    for (i = 0; i < heightCargas; i++) {
        fscanf(datosCargas, "%lf,%lf,%lf\n",&data->cargas[i*widthCargas],&data->cargas[i*widthCargas+1],&data->cargas[i*widthCargas+2]);
    }

    for (i = 0; i < heightGen; i++) {
        fscanf(datosGen, "%lf,%lf,%lf\n",&data->gen[i*widthGen],&data->gen[i*widthGen+1],&data->gen[i*widthGen+2]);
    }

    fclose(datosLineas);
    fclose(datosGen);
    fclose(datosCargas);
    return 0;
}

int printDataToFileVec(char *name, int size,double *data){
    FILE *dato;
    dato = fopen(name,"w");
    int i;
    for (i = 0; i < size; i++) {
        fprintf(dato,"%.4lf\n",data[i]);
    }
    fclose(dato);
    return 0;

}

int printDataToFileMat(char *name, int size,double *data){
    FILE *dato;
    dato = fopen(name,"w");
    int i,j;
    for (i = 0; i < size; i++) {
        for (j = 0; j < size; j++) {
            if(j!=size-1)
                fprintf(dato,"%.4lf ",data[i*size+j]);
            else
                fprintf(dato,"%.4lf\n",data[i*size+j]);
        }

    }

    fclose(dato);
    return 0;

}

int calcularMatrizA(structData *data, int widthLineas, double *A){
    zeros((data->numL)*(data->numN),A);
    int i;
    int N1,N2;
    for(i=0;i<(data->numL);i++){
        N1 = (int)(data->lineas[i*widthLineas+0])-1;
        N2 = (int)(data->lineas[i*widthLineas+1])-1;
        A[i*(int)(data->numN)+N1] = 1;
        A[i*(int)(data->numN)+N2] = -1;
    }
    return 0;
}

int printMatrixToFile(double *A, int numFilas, int numColumnas, char *name){
    FILE *dato;
    dato = fopen(name,"w");
    int i,j;
    for (i = 0; i < numFilas; i++) {
        for (j = 0; j < numColumnas; j++) {
            if(j!=numColumnas-1)
                fprintf(dato,"%.0lf ", A[i*(int)(numColumnas)+j]);
            else
                fprintf(dato,"%.0lf\n", A[i*(int)(numColumnas)+j]);

        }
    }
    fclose(dato);
    return 0;
}

int calcularZp(structData *data, int heightLineas, int widthLineas, double *ZpReal, double *ZpImag){
    int i;
    for (i = 0; i < heightLineas; i++) {
        ZpReal[i] = data->lineas[i*widthLineas+2];
        ZpImag[i] = data->lineas[i*widthLineas+3];

    }
    return 0;
}


int loadNW(char *fileNameNW, double *NW){
    FILE *datosNW;
    int numDataNW, i;
    numDataNW = 3;
    datosNW = fopen(fileNameNW,"r");
    if (datosNW == NULL){
        printf("Archivo de NW inexistente %s verifique \n",fileNameNW);
        exit(1);
    }

    for(i=0;i<numDataNW;i++){
        fscanf(datosNW,"%lf,%lf,%lf,%lf\n",&NW[i*4+0],&NW[i*4+1],&NW[i*4+2],&NW[i*4+3]);
    }

    fclose(datosNW);
}

int newtonRaphsonCUDA(structData *data, double *Vn, double *ybusReal, double *ybusImag){
    int res,i,j;
    int widthCargas = 3;
    int widthGen = 3;
    int widthLineas = 6;
    int heightCargas = 83;
    int heightGen = 15;
    int heightLineas = 186, NumP;
    ones(data->numN,Vn);
    zeros(data->numN,An);
    calcularYbus(data,ybusReal,ybusImag);
    NumP = (int) data->numN - 1;
    int *NNP = (int *)malloc((data->numN-1)*sizeof(int));
    genVector(NNP, 2,data->numN);
    int *vector1 = (int*)malloc(data->numN*sizeof(int));
    genVector(vector1, 1, data->numN);
    int *NNQ = (int *) malloc(data->numN*sizeof(int));
    int NumQ = setdiff(vector1, data->gen, data->numN, data->numG, NNQ);
    double *Pref = (double*)malloc(data->numN*sizeof(double));
    double *Qref = (double*)malloc(data->numN*sizeof(double));
    zeros(data->numN,Pref);
    zeros(data->numN,Qref);
    int k;
    int N1;

    for (k = 0; k < data->numG; k++) {
        N1 = (int) data->gen[k*widthGen+0] - 1;
        Pref[N1] = Pref[N1] + data->gen[k*widthGen+1];
        Vn[N1] = data->gen[k*widthGen+2];
    }

    for (k = 0 ; k < data->numC; k++) {
        N1 = (int)data->cargas[k*widthCargas] - 1;
        Pref[N1] = Pref[N1] - data->cargas[k*widthCargas+1];
        Qref[N1] = Qref[N1] - data->cargas[k*widthCargas+2];
    }


    double *dP = (double*)malloc(NumP*sizeof(double));
    double *dQ = (double*)malloc(NumQ*sizeof(double));

    zeros(NumP,dP);
    zeros(NumQ,dQ);

    double Error = 100.0;
    int iter = 0;
    int lda = NumP+NumQ,kk;
    int NumPQ = NumP+NumQ, nrhs = 1;
    int *ipiv,ldb = NumQ+NumP,info;

    double *Jpp, *Jpq, *Jqp, *Jqq, *Pn, *Qn, *JacR, *dPdQ, *JacRt,*dX, *Ism;
    double *d_Jpp, *d_Jpq, *d_Jqp, *d_Jqq, *d_Pn, *d_Qn, *d_ybusReal, *d_ybusImag, *d_Vn, *d_An;
    double *d_dX, *d_JacRt, *d_work, *d_dP, *d_dQ, *d_Pref, *d_Qref, *d_JacR, *d_dPdQ;
    int *devIpiv, *d_NNP, *d_NNQ;

    Jpp = (double*)malloc(data->numN*data->numN*sizeof(double));
    Jpq = (double*)malloc(data->numN*data->numN*sizeof(double));
    Jqp = (double*)malloc(data->numN*data->numN*sizeof(double));
    Jqq = (double*)malloc(data->numN*data->numN*sizeof(double));
    Pn = (double*)malloc(data->numN*sizeof(double));
    Qn = (double*)malloc(data->numN*sizeof(double));
    JacR = (double*)malloc((NumPQ)*(NumPQ)*sizeof(double));
    dPdQ = (double*)malloc((NumPQ)*sizeof(double));
    dX = (double*)malloc((NumPQ)*sizeof(double));
    ipiv = (int*)malloc((NumP+NumQ)*sizeof(int));
    JacRt = (double*)malloc((NumPQ)*(NumPQ)*sizeof(double));
    Ism = (double*)malloc(data->numL*sizeof(double));

    hipError_t error = hipSuccess;
    int lwork =0, *devInfo;
    gpuErrchk(hipMalloc(&devInfo, sizeof(int)));


    gpuErrchk(hipMalloc(&d_ybusImag,data->numN*data->numN*sizeof(double)));
    gpuErrchk(hipMalloc(&d_ybusReal,data->numN*data->numN*sizeof(double)));
    gpuErrchk(hipMalloc(&d_Qn,data->numN*sizeof(double)));
    gpuErrchk(hipMalloc(&d_Pn,data->numN*sizeof(double)));
    gpuErrchk(hipMalloc(&d_Jpp,data->numN*data->numN*sizeof(double)));
    gpuErrchk(hipMalloc(&d_Jqp,data->numN*data->numN*sizeof(double)));
    gpuErrchk(hipMalloc(&d_Jpq,data->numN*data->numN*sizeof(double)));
    gpuErrchk(hipMalloc(&d_Jqq,data->numN*data->numN*sizeof(double)));
    gpuErrchk(hipMalloc(&d_Vn,data->numN*sizeof(double)));
    gpuErrchk(hipMalloc(&d_An,data->numN*sizeof(double)));
    gpuErrchk(hipMalloc(&d_dP,NumP*sizeof(double)));
    gpuErrchk(hipMalloc(&d_dQ,NumQ*sizeof(double)));
    gpuErrchk(hipMalloc(&d_Pref,(data->numN)*sizeof(double)));
    gpuErrchk(hipMalloc(&d_Qref,(data->numN)*sizeof(double)));
    gpuErrchk(hipMalloc(&d_NNP,((data->numN)-1)*sizeof(int)));
    gpuErrchk(hipMalloc(&d_NNQ,(data->numN)*sizeof(int)));
    gpuErrchk(hipMalloc(&d_JacR,NumPQ*NumPQ*sizeof(double)));
    gpuErrchk(hipMalloc(&d_dPdQ,NumPQ*sizeof(double)));


    // ---- Copy ybusData to GPU ----//
    gpuErrchk(hipMemcpy(d_ybusReal,ybusReal,sizeof(double)*data->numN*data->numN\
                ,hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_ybusImag,ybusImag,sizeof(double)*data->numN*data->numN\
                ,hipMemcpyHostToDevice));
    /*---- Copy Pref and Qref to device ----*/
    gpuErrchk(hipMemcpy(d_Pref,Pref,sizeof(double)*data->numN,hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_Qref,Qref,sizeof(double)*data->numN,hipMemcpyHostToDevice));

    /*---- Copy Pref and Qref to device ----*/
    gpuErrchk(hipMemcpy(d_NNP,NNP,sizeof(int)*((data->numN)-1),hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_NNQ,NNQ,sizeof(int)*data->numN,hipMemcpyHostToDevice));
    /*---- Copy An y Vn al device -----*/
    gpuErrchk(hipMemcpy(d_Vn,Vn,sizeof(double)*data->numN,hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_An,An,sizeof(double)*data->numN,hipMemcpyHostToDevice));


    // ---- cuSolver initialization ---- //
    hipsolverStatus_t solvStatus = HIPSOLVER_STATUS_SUCCESS;
    hipsolverHandle_t handle;
    solvStatus = hipsolverDnCreate(&handle);
    ///////////////////////////////////////

    hipblasOperation_t trans = HIPBLAS_OP_N;


    gpuErrchk(hipMalloc((void**)&d_JacRt,sizeof(double)*NumPQ*NumPQ));
    gpuErrchk(hipMalloc((void**)&d_dX,sizeof(double)*NumPQ));
    gpuErrchk(hipMalloc((void**)&devIpiv,sizeof(int)*NumPQ));

    cusolveSafeCall(hipsolverDnDgetrf_bufferSize(handle,NumPQ,NumPQ,d_JacRt,lda,&lwork));

    gpuErrchk(hipMalloc((void**)&d_work, sizeof(double)*lwork));

    zeros(data->numL,Ism);

    int blockSize2D = 32;
    int blockSize = 1024;
    dim3 dimBlock(blockSize,1,1);
    dim3 dimGrid(ceil(data->numN/float(blockSize)),1,1);

    dim3 dimBlockXY(blockSize2D,blockSize2D,1);
    dim3 dimGridXY(ceil(data->numN/float(blockSize2D)),ceil(data->numN/float(blockSize2D)),1);

    dim3 dimBlock2(blockSize2D,blockSize2D,1);
    dim3 dimGrid2(ceil(data->numN/float(blockSize2D)),ceil(data->numN/float(blockSize2D)),1);

    dim3 dimGrid3(ceil((data->numN*data->numN)/float(blockSize)),1,1);
    dim3 dimGrid4(ceil(NumP/float(blockSize)),1,1);
    dim3 dimGrid5(ceil(NumQ/float(blockSize)),1,1);
    dim3 dimGrid6(ceil(NumP/float(blockSize2D)), ceil(NumP/float(blockSize2D)),1);
    dim3 dimGrid7(ceil(NumP/float(blockSize)),1,1);
    dim3 dimGrid8(ceil(NumPQ/float(blockSize2D)),ceil(NumPQ/float(blockSize2D)),1);
    dim3 dimGrid9(ceil(NumQ/float(blockSize)),1,1);
    dim3 dimGrid10(ceil(NumPQ/float(blockSize)),1,1);

    while (Error>1e-8){

        /*---- Initialize d_Jpp, d_Jpq, d_Jqp, d_Jqq, ----*/
        d_zeros<<<dimGrid3,dimBlock>>>(data->numN*data->numN,d_Jpp);
        d_zeros<<<dimGrid3,dimBlock>>>(data->numN*data->numN,d_Jpq);
        d_zeros<<<dimGrid3,dimBlock>>>(data->numN*data->numN,d_Jqp);
        d_zeros<<<dimGrid3,dimBlock>>>(data->numN*data->numN,d_Jqq);
        hipDeviceSynchronize();

        d_calcularJacobiano_1<<<dimGrid,dimBlock>>>(data->numN, d_ybusReal, d_ybusImag,d_Vn,\
                d_An,d_Pn,d_Qn);

        hipDeviceSynchronize();
        d_calcularJacobiano_2<<<dimGrid2,dimBlock2>>>(data->numN, d_ybusReal, d_ybusImag, d_Vn\
                ,d_An, d_Pn,d_Qn, d_Jpp, d_Jpq, d_Jqp, d_Jqq);
        hipDeviceSynchronize();

        dp_compute<<<dimGrid4,dimBlock>>>(NumP, d_NNP, d_Pref, d_Pn, d_dP);

        dq_compute<<<dimGrid5,dimBlock>>>(NumP, d_NNQ, d_Qref, d_Qn, d_dQ);

        d_createJacR_1<<<dimGrid6,dimBlock2>>>(d_NNP, NumQ, NumP, (int)(data->numN), d_Jpp, d_JacR);
        hipDeviceSynchronize();
        d_createJacR_2<<<dimGrid7,dimBlock>>>(d_NNP, d_NNQ, NumQ, NumP, (int)(data->numN), d_Jpq, \
                d_JacR);
        hipDeviceSynchronize();
        d_createJacR_3<<<dimGrid7,dimBlock>>>(d_NNP, d_NNQ, NumQ, NumP,(int)(data->numN), d_Jqp, \
                d_JacR);
        hipDeviceSynchronize();
        d_createJacR_4<<<dimGrid5,dimBlock>>>(d_NNQ, NumQ, NumP,(int)(data->numN),\
                d_Jqq, d_JacR);
        hipDeviceSynchronize();


        d_transposeJacr<<<dimGrid8,dimBlock2>>>(d_JacR, NumPQ, d_JacRt);
        hipDeviceSynchronize();

        d_filldPdQ1<<<dimGrid7,dimBlock>>>(d_dP,NumP,d_dPdQ);
        hipDeviceSynchronize();
        d_filldPdQ<<<dimGrid9,dimBlock>>>(d_dQ,NumQ,NumP,d_dPdQ);
        hipDeviceSynchronize();

        d_fill_d_dx<<<dimGrid10,dimBlock>>>(d_dPdQ,NumPQ,d_dX);
        hipDeviceSynchronize();


        cusolveSafeCall(hipsolverDnDgetrf(handle, NumPQ, NumPQ,d_JacRt,NumPQ,d_work,devIpiv,devInfo));
        gpuErrchk(hipDeviceSynchronize());
        cusolveSafeCall(hipsolverDnDgetrs(handle,trans,NumPQ,nrhs,d_JacRt,NumPQ,devIpiv,d_dX,NumPQ,\
                    devInfo));
        hipDeviceSynchronize();


        d_calc_An<<<dimGrid7,dimBlock>>>(d_dX, d_NNP, NumP, d_An);
        hipDeviceSynchronize();

        d_calc_Vn<<<dimGrid9,dimBlock>>>(d_dX, d_NNQ, NumP, NumQ, d_Vn);
        hipDeviceSynchronize();

        gpuErrchk(hipMemcpy(dPdQ, d_dPdQ,sizeof(double)*NumPQ,hipMemcpyDeviceToHost));
        Error = maxAbs(NumPQ,dPdQ);

        if (iter>data->maxIter) {
            printf("..... No converge despues de %d iteraciones\nError = %lf\n", data->maxIter, Error);
            break;
        }
        iter++;
    }


    gpuErrchk(hipMemcpy(An,d_An,sizeof(double)*data->numN,hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(Vn,d_Vn,sizeof(double)*data->numN,hipMemcpyDeviceToHost));
    calcCargLineas(data,An,Vn,Ism);
    printDataToFileVec("ismData",data->numL,Ism);
    printDataToFileVec("vnData",data->numN,Vn);
    printDataToFileVec("anData",data->numN,An);

    hipsolverDnDestroy(handle);
    free(An);
    free(NNP);
    free(NNQ);
    free(vector1);
    free(dP);
    free(dQ);
    free(Pref);
    free(Qref);
    free(JacR);
    free(ipiv);
    free(JacRt);
    free(dX);
    free(Jpp);
    free(Jpq);
    free(Jqp);
    free(Jqq);
    free(Ism);
    hipFree(d_ybusReal);
    hipFree(d_ybusImag);
    hipFree(d_work);
    hipFree(d_JacRt);
    hipFree(devIpiv);
    hipFree(d_work);
    hipFree(d_Jpp);
    hipFree(d_Jqq);
    hipFree(d_Jpq);
    hipFree(d_Jqp);
    hipFree(d_Vn);
    hipFree(d_An);
    hipFree(d_Pref);
    hipFree(d_Qref);
    hipFree(d_dP);
    hipFree(d_dQ);
    hipFree(d_JacR);
    hipFree(d_JacRt);
    hipFree(d_dPdQ);
    return res;
}

int newtonRaphson(structData *data, double *Vn, double *An, double *ybusReal, double *ybusImag){
    int res,i,j;
    int widthCargas = 3;
    int widthGen = 3;
    int widthLineas = 6;
    int heightCargas = 83;
    int heightGen = 15;
    int heightLineas = 186, NumP;
    //Vn = (double*)malloc(data->numN*sizeof(double));
    //An = (double*)malloc(data->numN*sizeof(double));
//    double *ybusReal = (double*) malloc(data->numN*data->numN*sizeof(double));
  //  double *ybusImag = (double*) malloc(data->numN*data->numN*sizeof(double));
    ones(data->numN,Vn);
    zeros(data->numN,An);
    calcularYbus(data,ybusReal,ybusImag);
    NumP = (int) data->numN - 1;
    int *NNP = (int *)malloc((data->numN-1)*sizeof(int));
    genVector(NNP, 2,data->numN);
    int *vector1 = (int*)malloc(data->numN*sizeof(int));
    genVector(vector1, 1, data->numN);
    int *NNQ = (int *) malloc(data->numN*sizeof(int));
    int NumQ = setdiff(vector1, data->gen, data->numN, data->numG, NNQ);
    double *Pref = (double*)malloc(data->numN*sizeof(double));
    double *Qref = (double*)malloc(data->numN*sizeof(double));
    zeros(data->numN,Pref);
    zeros(data->numN,Qref);
    int k;
    int N1;

    for (k = 0; k < data->numG; k++) {
        N1 = (int) data->gen[k*widthGen+0] - 1;
        Pref[N1] = Pref[N1] + data->gen[k*widthGen+1];
        Vn[N1] = data->gen[k*widthGen+2];
    }

    for (k = 0 ; k < data->numC; k++) {
        N1 = (int)data->cargas[k*widthCargas] - 1;
        Pref[N1] = Pref[N1] - data->cargas[k*widthCargas+1];
        Qref[N1] = Qref[N1] - data->cargas[k*widthCargas+2];
    }

    double *dP = (double*)malloc(NumP*sizeof(double));
    double *dQ = (double*)malloc(NumQ*sizeof(double));

    zeros(NumP,dP);
    zeros(NumQ,dQ);

    double Error = 100.0;
    int iter = 0;
    int lda = NumP+NumQ,kk;
    int NumPQ = NumP+NumQ, nrhs = 1;
    int *ipiv,ldb = NumQ+NumP,info;

    double *Jpp, *Jpq, *Jqp, *Jqq, *Pn, *Qn, *JacR, *dPdQ, *JacRt,*dX, *Ism;

    Jpp = (double*)malloc(data->numN*data->numN*sizeof(double));
    Jpq = (double*)malloc(data->numN*data->numN*sizeof(double));
    Jqp = (double*)malloc(data->numN*data->numN*sizeof(double));
    Jqq = (double*)malloc(data->numN*data->numN*sizeof(double));
    Pn = (double*)malloc(data->numN*sizeof(double));
    Qn = (double*)malloc(data->numN*sizeof(double));
    JacR = (double*)malloc((NumPQ)*(NumPQ)*sizeof(double));
    dPdQ = (double*)malloc((NumPQ)*sizeof(double));
    dX = (double*)malloc((NumPQ)*sizeof(double));
    ipiv = (int*)malloc((NumP+NumQ)*sizeof(int));
    JacRt = (double*)malloc((NumPQ)*(NumPQ)*sizeof(double));
    Ism = (double*)malloc(data->numL*sizeof(double));

    zeros(data->numL,Ism);
    while (Error>1e-8){
        calcularJacobiano(data,ybusReal,ybusImag,Vn,An,Jpp,Jpq,Jqp,Jqq,Pn,Qn);

        for (i = 0 ; i < NumP ; i++) {
            N1 = NNP[i] - 1;
            dP[i] = Pref[N1] - Pn[N1];
        }

        for (i = 0; i < NumQ; i++ ) {
            N1 = NNQ[i] - 1;
            dQ[i] = Qref[N1] - Qn[N1];
        }

        createJacR(NNP, NNQ, NumQ, NumP, (int)data->numN, Jpp, Jpq, Jqp, Jqq, JacR);
        transposeJacR(JacR,NumPQ,JacRt);
        createdPdQ(dP,dQ,NumP,NumQ,dPdQ);
        memcpy(dX,dPdQ,sizeof(double)*NumPQ);
        dgesv_(&NumPQ,&nrhs,JacRt,&lda,ipiv,dX,&ldb,&info);

        for (k = 0; k < NumP; k++) {
            N1 = NNP[k] - 1;
            An[N1] = An[N1] + dX[k];
        }

        for (k = 0; k < NumQ; k++) {
            N1 = NNQ[k] - 1;
            kk = k + NumP;
            Vn[N1] = Vn[N1] + dX[kk];
        }

       Error = maxAbs(NumPQ,dPdQ);

        if (iter>data->maxIter) {
            printf("..... No converge despues de %d iteraciones\nError = %lf\n", data->maxIter, Error);
            break;
        }
        iter++;
    }

    calcCargLineas(data,An,Vn,Ism);
    //printDataToFileVec("ismData",data->numL,Ism);
    //printDataToFileVec("vnData",data->numN,Vn);
    //printDataToFileVec("anData",data->numN,An);

    //free(ybusReal);
    //free(ybusImag);
    free(NNP);
    free(NNQ);
    free(vector1);
    free(dP);
    free(dQ);
    free(Pref);
    free(Qref);
    free(JacR);
    free(ipiv);
    free(JacRt);
    free(dX);
    free(Jpp);
    free(Jpq);
    free(Jqp);
    free(Jqq);
    free(Ism);
    return res;
}

int calculoIrama(double *Vrama, double *ZpReal, double *ZpImag, int heightLineas, double *Irama){
    int i;
    for (i = 0; i < heightLineas; i++) {
        Irama[i] = fabs(Vrama[i]/ZpReal[i]);
    }
    return 0;
}


int calculoVn(double *Vn, double *An, int height, double *VnReal, double *VnImag){
    int i;
    for (i = 0; i < height; i++) {
        VnReal[i] = Vn[i]*cos(An[i]);
        VnImag[i] = Vn[i]*sin(An[i]);
    }
    return 0;
}

int addVectors(double *vec1, double *vec2, int numN){
    int i;
    for (i = 0; i < numN; i++) {
        vec1[i] = vec1[i] + vec2[i];
    }
    return 0;
}

int subVectors(double *vec1, double *vec2, int numN){
    int i;
    for (i = 0; i < numN; i++) {
        vec1[i] = vec1[i] - vec2[i];
    }
    return 0;
}

int calculoSobrecarga(double *IlineaReal, double *IlineaImag, double *sobrecarga, double *Imax, int numL){
    int i;
    for (i = 0; i < numL; i++) {
        sobrecarga[i] = sqrt(IlineaReal[i]*IlineaReal[i] + IlineaImag[i]*IlineaImag[i])/Imax[i];
    }
    return 0;
}

int calculoMontSobrecarga(int numL, double *sobrecarga, Mont *mont){
    int i;
    for (i = 0; i < numL; i++) {
        if(sobrecarga[i]>1.0)
            mont->sob[i] = mont->sob[i] + 1;
    }
    return 0;
}


int calculoCorrientesRama(int numN,double *Vn,Mont *mont){
    int i;
    for (i = 0; i < numN; i++) {
        mont->sum[i] = mont->sum[i] + Vn[i];
        mont->sumcuad[i] = mont->sumcuad[i] + Vn[i] * Vn[i];
        if(Vn[i]<0.9)
            mont->lv[i] = mont->lv[i] + 1;
        if(Vn[i]>1.1)
            mont->hv[i] = mont->hv[i] + 1;
    }
    return 0;
}

int calculosFinales(int numN,int ni, Mont *mont, double *Vmedia, double *Vdesv, double *Probmin, double *ProbMax){
    int i;
    for (i = 0; i < numN; i++) {
        Vmedia[i] = mont->sum[i]/ni;
        Vdesv[i] = mont->sumcuad[i] - 2*mont->sum[i]*Vmedia[i] - Vmedia[i]*Vmedia[i];
        Probmin[i] = mont->lv[i]/ni;
        ProbMax[i] = mont->hv[i]/ni;
    }
}

int calculoProbSobrecarga(int numL, int ni, Mont *mont, double *Probsobrecarga){
    int i;
    for (i = 0; i < numL; i++) {
        Probsobrecarga[i] = mont->sob[i]/ni;
    }
    return 0;
}
