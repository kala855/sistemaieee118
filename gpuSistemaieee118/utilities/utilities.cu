#include "hip/hip_runtime.h"
#include <stdlib.h>
#include "utilities.h"
#include <string.h>
#include <math.h>


int calcCargLineas(structData *data, double *An, double *Vn, double *Ism){
    int k, N1, N2;
    double akm;
    int widthLineas = 6;
    for (k = 0; k < data->numL; k++) {
        N1 = data->lineas[k*widthLineas+0] - 1;
        N2 = data->lineas[k*widthLineas+1] - 1;
        akm = An[N1] - An[N2];
        Ism[k] = sqrt(Vn[N1]*Vn[N1] + Vn[N2]*Vn[N2] - 2*Vn[N1]*Vn[N2]*cos(akm));
        Ism[k] = Ism[k]/sqrt(data->lineas[k*widthLineas+2]*data->lineas[k*widthLineas+2] + \
                data->lineas[k*widthLineas+3]*data->lineas[k*widthLineas+3])\
                 /1.6;//data->lineas[k*widthLineas+6];

    }
    return 0;
}

double maxAbs(int NumPQ, double *dPdQ){
    int i;
    double max = 0.0;
    for (i = 0; i < NumPQ; i++){
        if(fabs(dPdQ[i]) > max)
            max = fabs(dPdQ[i]);
    }
    return max;
}

int transposeJacR(double *JacR,int NumPQ, double *JacRt){
    int i, j;
    for ( i = 0; i < NumPQ; i++) {
        for (j = 0; j < NumPQ; j++) {
            JacRt[i*NumPQ+j] = JacR[j*NumPQ+i];
        }
    }
    return 0;

}

int createdPdQ(double *dP, double *dQ, int NumP, int NumQ, double *dPdQ){
    memcpy(dPdQ,dP,sizeof(double)*NumP);
    memcpy(dPdQ+NumP,dQ,sizeof(double)*NumQ);
    return 0;
}



__global__ void d_calc_An(double *dX, int *NNP, int NumP, double *An){
    int k = blockIdx.x*blockDim.x+threadIdx.x;
    int N1;
    if (k<NumP) {
        N1 = NNP[k] - 1;
        An[N1] = An[N1] + dX[k];
    }
}

__global__ void d_calc_Vn(double *dX, int *NNQ, int NumP, int NumQ, double *Vn){
    int k = blockIdx.x*blockDim.x+threadIdx.x;
    int kk, N1;
    if (k<NumQ) {
        N1 = NNQ[k] - 1;
        kk = k + NumP;
        Vn[N1] = Vn[N1] + dX[kk];
    }
}


__global__ void d_fill_d_dx(double *d_dPdQ, int NumPQ, double *d_dX){
    int i = blockIdx.x*blockDim.x+threadIdx.x;
    if(i<NumPQ){
        d_dX[i] = d_dPdQ[i];
    }
}


__global__ void d_filldPdQ1(double *d_dP, int NumP, double *d_dPdQ){
    int i = blockIdx.x*blockDim.x+threadIdx.x;
    if(i<NumP){
        d_dPdQ[i] = d_dP[i];
    }
}



__global__ void d_filldPdQ(double *d_dQ, int NumQ, int NumP, double *d_dPdQ){
    int i = blockIdx.x*blockDim.x+threadIdx.x;
    if(i<NumQ){
        d_dPdQ[i+NumP] = d_dQ[i];
    }
}

__global__ void d_transposeJacr(double *JacR,int NumPQ, double *JacRt){
    int i = blockIdx.y*blockDim.y+threadIdx.y;
    int j = blockIdx.x*blockDim.x+threadIdx.x;
    if(i<NumPQ && j <NumPQ)
        JacRt[i*NumPQ+j] = JacR[j*NumPQ+i];

}

__global__ void d_zeros2(int size,double *An){
    int i = blockIdx.y*blockDim.y+threadIdx.y;
    int j = blockIdx.x*blockDim.x+threadIdx.x;
    if(i < size && j < size) {
        An[i*size+j] = 0.0;
    }
}


__global__ void d_zeros(int size,double *An){
    int i = blockIdx.x*blockDim.x+threadIdx.x;
    if(i < size) {
        An[i] = 0.0;
    }
}

__global__ void d_ones(int size,double *Vn){
    int i = blockIdx.x*blockDim.x+threadIdx.x;
    if( i < size){
        Vn[i] = 1.0;
    }
}

__global__ void dp_compute(int NumP,int *NNP, double *Pref, double *Pn, double *dP){
    int i = blockIdx.x*blockDim.x+threadIdx.x;
    int N1;
    if(i<NumP){
        N1 = NNP[i] - 1;
        dP[i] = Pref[N1] - Pn[N1];
    }
}

__global__ void dq_compute(int NumQ,int *NNQ, double *Qref, double *Qn,double *dQ){
    int i = blockIdx.x*blockDim.x+threadIdx.x;
    int N1;
    if(i<NumQ){
        N1 = NNQ[i] - 1;
        dQ[i] = Qref[N1] - Qn[N1];
    }
}



__global__ void d_createJacR_1(int *NNP, int NumQ, int NumP, int numN, double *Jpp, double *JacR){
    int i = blockIdx.y*blockDim.y+threadIdx.y;
    int j = blockIdx.x*blockDim.x+threadIdx.x;
    int size = NumP+NumQ;
    if((i<NumP) && (j<NumP)){
        JacR[i*size+j] =  Jpp[(NNP[i]-1)*numN+(NNP[j]-1)];
    }
}


__global__ void d_createJacR_2(int *NNP, int *NNQ, int NumQ, int NumP, int numN, double *Jpq, \
        double *JacR){
    int i = blockIdx.x*blockDim.x+threadIdx.x;
    int size = NumP+NumQ;
    int j = 0;
    int k = i,l;
    if(i<NumP){
        l = NumP;
        for(j=0; j<NumQ; j++){
            JacR[k*size+l] = Jpq[(NNP[i]-1)*numN+(NNQ[j]-1)];
            l++;
        }
        //k++;
    }
}



__global__ void d_createJacR_3(int *NNP, int *NNQ, int NumQ, int NumP, int numN,double *Jqp, \
        double *JacR){
    int i = blockIdx.x*blockDim.x+threadIdx.x;
    int j = 0;
    int k = NumP+i,l;
    int size = NumP+NumQ;
    if(i<NumQ){
        l = 0;
        for(j=0; j<NumP; j++){
            JacR[k*size+l] = Jqp[(NNQ[i]-1)*numN+(NNP[j]-1)];
            l++;
        }
        //k++;
    }
}



__global__ void d_createJacR_4(int *NNQ, int NumQ, int NumP, int numN, double *Jqq, double *JacR){
    int i = blockIdx.x*blockDim.x+threadIdx.x;
    int size = NumP+NumQ;
    int j = 0;
    int k = NumP+i,l;
    if(i<NumQ){
        l = NumP;
        for(j=0; j<NumQ; j++){
            JacR[k*size+l] = Jqq[(NNQ[i]-1)*numN+(NNQ[j]-1)];
            l++;
        }
    }
}

/*Esta funcion permite actualizar los valores de Pn y Qn que posteriormente serán usados
 para el cálculo completo del Jacobiano*/
__global__ void d_calcularJacobiano_1(int numN, double *ybusReal, double *ybusImag, double *Vn, \
        double *An, double *Pn, double *Qn){
    int k = blockIdx.x*blockDim.x+threadIdx.x;
    int widthLineas = numN;
    int m;
    double akm;
    if (k<numN){
        Pn[k] = 0.0;
        Qn[k] = 0.0;
        for (m = 0; m < numN; m++){
            akm = An[k] - An[m];
            Pn[k] = Pn[k] + Vn[m]*(ybusReal[k*widthLineas+m]*cos(akm) + \
                    ybusImag[k*widthLineas+m]*sin(akm));
            Qn[k] = Qn[k] - Vn[m]*(ybusImag[k*widthLineas+m]*cos(akm) - \
                    ybusReal[k*widthLineas+m]*sin(akm));

        }
        Pn[k] = Pn[k]*Vn[k];
        Qn[k] = Qn[k]*Vn[k];
    }


}

__global__ void d_calcularJacobiano_2(int numN, double *ybusReal, double *ybusImag, double *Vn,\
        double *An,double *Pn, double *Qn, double *Jpp, double *Jpq, double *Jqp, double *Jqq){

    int k = blockIdx.y*blockDim.y+threadIdx.y;
    int m = blockIdx.x*blockDim.x+threadIdx.x;
    int widthLineas = numN;
    double akm;
    if(k<numN){
        if(m<numN)
            /*Jpp[k*widthLineas+m] = 0.0;
            Jpq[k*widthLineas+m] = 0.0;
            Jqp[k*widthLineas+m] = 0.0;
            Jqq[k*widthLineas+m] = 0.0;*/

            if(k==m){
                Jpp[k*widthLineas+k] = -ybusImag[k*widthLineas+k]*Vn[k]*Vn[k]-Qn[k];
                Jpq[k*widthLineas+k] = ybusReal[k*widthLineas+k]*Vn[k] + Pn[k]/Vn[k];
                Jqp[k*widthLineas+k] = -ybusReal[k*widthLineas+k]*Vn[k]*Vn[k] + Pn[k];
                Jqq[k*widthLineas+k] = -ybusImag[k*widthLineas+k]*Vn[k] + Qn[k]/Vn[k];
            }else{
                akm = An[k] - An[m];
                Jpp[k*widthLineas+m] = Vn[k]*Vn[m]*(ybusReal[k*widthLineas+m]*sin(akm)- \
                        ybusImag[k*widthLineas+m]*cos(akm)) ;
                Jpq[k*widthLineas+m] = Vn[k]*(ybusReal[k*widthLineas+m]*cos(akm)+ybusImag[k*widthLineas+m]*sin(akm));
                Jqp[k*widthLineas+m] = -Jpq[k*widthLineas+m]*Vn[m];
                Jqq[k*widthLineas+m] = Jpp[k*widthLineas+m]/Vn[m];
            }
    }

}

int calcularJacobiano(structData *data, double *ybusReal, double *ybusImag, double *Vn, double *An, \
        double *Jpp, double *Jpq, double *Jqp, double *Jqq, double *Pn, double *Qn){

    zeros(data->numN,Pn);
    zeros(data->numN,Qn);
    double akm, *H,*N,*J,*L;
    int k,m,widthLineas=data->numN;
    for (k = 0; k < data->numN; k++) {
        for (m = 0; m < data->numN; m++) {
            akm = An[k] - An[m];
            Pn[k] = Pn[k] + Vn[m]*(ybusReal[k*widthLineas+m]*cos(akm) + \
                    ybusImag[k*widthLineas+m]*sin(akm));
            Qn[k] = Qn[k] - Vn[m]*(ybusImag[k*widthLineas+m]*cos(akm) - \
                    ybusReal[k*widthLineas+m]*sin(akm));
        }
        Pn[k] = Pn[k]*Vn[k];
        Qn[k] = Qn[k]*Vn[k];
    }

    H = (double*)malloc(data->numN*data->numN*sizeof(double));
    N = (double*)malloc(data->numN*data->numN*sizeof(double));
    J = (double*)malloc(data->numN*data->numN*sizeof(double));
    L = (double*)malloc(data->numN*data->numN*sizeof(double));

    zeros(data->numN*data->numN,H);
    zeros(data->numN*data->numN,N);
    zeros(data->numN*data->numN,J);
    zeros(data->numN*data->numN,L);

    for (k = 0; k < data->numN; k++) {
        for (m = 0; m < data->numN; m++) {
            if(k==m){
                H[k*widthLineas+k] = -ybusImag[k*widthLineas+k]*Vn[k]*Vn[k]-Qn[k];
                N[k*widthLineas+k] = ybusReal[k*widthLineas+k]*Vn[k] + Pn[k]/Vn[k];
                J[k*widthLineas+k] = -ybusReal[k*widthLineas+k]*Vn[k]*Vn[k] + Pn[k];
                L[k*widthLineas+k] = -ybusImag[k*widthLineas+k]*Vn[k] + Qn[k]/Vn[k];
            }else{
                akm = An[k] - An[m];
                H[k*widthLineas+m] = Vn[k]*Vn[m]*(ybusReal[k*widthLineas+m]*sin(akm)- \
                        ybusImag[k*widthLineas+m]*cos(akm)) ;
                N[k*widthLineas+m] = Vn[k]*(ybusReal[k*widthLineas+m]*cos(akm)+ybusImag[k*widthLineas+m]*sin(akm));
                J[k*widthLineas+m] = -N[k*widthLineas+m]*Vn[m];
                L[k*widthLineas+m] = H[k*widthLineas+m]/Vn[m];
            }
        }
    }

    memcpy(Jpp,H,data->numN*data->numN*sizeof(double));
    memcpy(Jpq,N,data->numN*data->numN*sizeof(double));
    memcpy(Jqp,J,data->numN*data->numN*sizeof(double));
    memcpy(Jqq,L,data->numN*data->numN*sizeof(double));

    free(H);free(N);free(J);free(L);
    return 0;

}

int setdiff(int *vector1, double *vector2, int size1, int size2, int *c){
    int i, j, accum=0, k=0;
    for (i = 0; i < size1; i++) {
        for (j = 0; j < size2; j++) {
            if(vector1[i]!=(int)vector2[j*3])
                accum++;
            else
                break;
        }
        if(accum == size2){
            c[k] = vector1[i];
            k++;
        }
        accum = 0;
    }
    return k;
}

int genVector(int *NNP, int initNumber,int finalNumber){
    int i = initNumber,j=0;
    for (i = initNumber; i <= finalNumber; i++) {
        NNP[j] = i;
        j++;
    }
    return 0;
}

int calcularYbus(structData *data, double *ybusReal, double *ybusImag){
    zeros(data->numN*data->numN, ybusReal);
    zeros(data->numN*data->numN, ybusImag);
    int widthLineas = 6;
    double N1, N2, ym;
    double YYReal, YYImag, tap;
    int k,t;
    for (k = 0; k < data->numL; k++) {
        N1 = data->lineas[k*widthLineas+0]-1;
        N2 = data->lineas[k*widthLineas+1]-1;
        ym = data->lineas[k*widthLineas+2]*data->lineas[k*widthLineas+2]+\
             data->lineas[k*widthLineas+3]*data->lineas[k*widthLineas+3];
        YYReal = data->lineas[k*widthLineas+2]/ym;
        YYImag = -data->lineas[k*widthLineas+3]/ym;

        tap = 1/data->lineas[k*widthLineas+5];

        ybusReal[(int) (N1*data->numN + N1)] = ybusReal[(int) (N1*data->numN + N1)]\
                                               + tap*tap*YYReal;
        ybusReal[(int) (N2*data->numN + N2)] = ybusReal[(int) (N2*data->numN + N2)] \
                                               + YYReal;
        ybusReal[(int) (N1*data->numN + N2)] = ybusReal[(int) (N1*data->numN + N2)] \
                                               - tap*YYReal;
        ybusReal[(int) (N2*data->numN + N1)] = ybusReal[(int) (N2*data->numN + N1)] \
                                               - tap*YYReal;

        ybusImag[(int) (N1*data->numN + N1)] = ybusImag[(int) (N1*data->numN + N1)] \
                                               + tap*tap*YYImag + data->lineas[k*widthLineas+4];
        ybusImag[(int) (N2*data->numN + N2)] = ybusImag[(int) (N2*data->numN + N2)] \
                                               + YYImag + data->lineas[k*widthLineas+4];
        ybusImag[(int) (N1*data->numN + N2)] = ybusImag[(int) (N1*data->numN + N2)] \
                                               - tap*YYImag;
        ybusImag[(int) (N2*data->numN + N1)] = ybusImag[(int) (N2*data->numN + N1)] \
                                               - tap*YYImag;
    }

    return 0;


}

int zeros(int size,double *An){
    int i;
    for (i = 0; i < size; i++) {
        An[i] = 0.0;
    }
    return 0;
}

int ones(int size,double *Vn){
    int i;
    for (i = 0; i < size; i++) {
        Vn[i] = 1.0;
    }
    return 0;
}

double maxLineas(structData *data, int widthLineas,int heightLineas){
    int i;
    double mayorLineas1 = 0, mayorLineas2;
    for (i = 0; i < heightLineas; i++) {
        if(mayorLineas1 < data->lineas[i*widthLineas]){
            mayorLineas1 = data->lineas[i*widthLineas];
        }
        if(mayorLineas2 < data->lineas[i*widthLineas+1]){
            mayorLineas2 = data->lineas[i*widthLineas+1];
        }
    }

    if(mayorLineas2>=mayorLineas1)
        return mayorLineas2;
    else
        return mayorLineas1;

}

int printData(structData *data, int widthLineas, int heightLineas, int widthCargas, int heightCargas, int widthGen, int heightGen){
    int i;
    printf("Datos Líneas\n");
    for (i = 0; i < heightLineas; i++) {
        printf("%.5lf,%.5lf,%.5lf,%.5lf,%.5lf\n",data->lineas[i*widthLineas],data->lineas[i*widthLineas+1],data->lineas[i*widthLineas+2],data->lineas[i*widthLineas+3],data->lineas[i*widthLineas+4]);
    }

    printf("Datos Cargas\n");
    for (i = 0; i < heightCargas; i++) {
        printf("%.5lf,%.5lf,%.5lf\n",data->cargas[i*widthCargas],data->cargas[i*widthCargas+1],data->cargas[i*widthCargas+2]);
    }

    printf("Datos Gen\n");
    for (i = 0; i < heightGen; i++) {
        printf("%.5lf,%.5lf,%.5lf\n",data->gen[i*widthGen],data->gen[i*widthGen+1],data->gen[i*widthGen+2]);
    }

    return 0;
}

int loadDataFromFile(char *filenameLineas, char *filenameCargas, char *filenameGen, structData *data){
    FILE *datosLineas,*datosGen,*datosCargas;
    int i, j, widthLineas,heightLineas,widthGen, heightGen,widthCargas, heightCargas;
    widthLineas = 6;
    heightLineas = 1896;/*186*/
    widthGen = 3;
    heightGen = 15*10;
    widthCargas = 3;
    heightCargas = 83*10;

    datosGen = fopen(filenameGen,"r");
    datosLineas = fopen(filenameLineas,"r");
    datosCargas = fopen(filenameCargas,"r");

    if (datosGen == NULL){
        printf("Archivo de Gen inexistente %s verifique \n", filenameGen);
        exit(1);
    }

    if (datosLineas == NULL){
        printf("Archivo de Lineas inexistente %s verifique \n", filenameLineas);
        exit(1);
    }

    if(datosCargas == NULL){
        printf("Archivo de Cargas inexistente %s verifique \n", filenameCargas);
        exit(1);
    }

    data->lineas = (double*)malloc(widthLineas*heightLineas*sizeof(double));
    data->cargas = (double*)malloc(widthCargas*heightCargas*sizeof(double));
    data->gen = (double*)malloc(widthGen*heightGen*sizeof(double));

    if (data->lineas == NULL){
        printf("Imposible asignar memoria a lineas\n");
        exit(1);
    }
    if (data->gen == NULL){
        printf("Imposible asignar memoria a gen\n");
        exit(1);
    }

    if (data->cargas == NULL){
        printf("Imposible asignar memoria a cargas\n");
        exit(1);
    }
    for (i = 0; i < heightLineas; i++) {//Se adiciona el tap constante en 1.0 para este caso
        fscanf(datosLineas, "%lf,%lf,%lf,%lf,%lf\n",&data->lineas[i*widthLineas],\
                &data->lineas[i*widthLineas+1],&data->lineas[i*widthLineas+2],\
                &data->lineas[i*widthLineas+3],&data->lineas[i*widthLineas+4]);
        if(data->lineas[i*widthLineas+4]==0)
            data->lineas[i*widthLineas+4] = 1.0;

        data->lineas[i*widthLineas+5] = 1.0;
    }

    for (i = 0; i < heightCargas; i++) {
        fscanf(datosCargas, "%lf,%lf,%lf\n",&data->cargas[i*widthCargas],&data->cargas[i*widthCargas+1],&data->cargas[i*widthCargas+2]);
    }

    for (i = 0; i < heightGen; i++) {
        fscanf(datosGen, "%lf,%lf,%lf\n",&data->gen[i*widthGen],&data->gen[i*widthGen+1],&data->gen[i*widthGen+2]);
    }

    fclose(datosLineas);
    fclose(datosGen);
    fclose(datosCargas);
    return 0;
}

int printDataToFileVec(char *name, int size,double *data){
    FILE *dato;
    dato = fopen(name,"w");
    int i;
    for (i = 0; i < size; i++) {
        fprintf(dato,"%.4lf\n",data[i]);
    }
    fclose(dato);
    return 0;

}

int printDataToFileMat(char *name, int size,double *data){
    FILE *dato;
    dato = fopen(name,"w");
    int i,j;
    for (i = 0; i < size; i++) {
        for (j = 0; j < size; j++) {
            if(j!=size-1)
                fprintf(dato,"%.4lf ",data[i*size+j]);
            else
                fprintf(dato,"%.4lf\n",data[i*size+j]);
        }

    }

    fclose(dato);
    return 0;

}
