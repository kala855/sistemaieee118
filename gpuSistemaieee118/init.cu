#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include "utilities/utilities.h"
#include <hipsolver.h>
#include "utilities/cusolverUtilities.cuh"


int main(){
    int res,i,j;
    int widthCargas = 3;
    int widthGen = 3;
    int widthLineas = 6;
    int heightCargas = 83;
    int heightGen = 15;
    int heightLineas = 186, NumP;
    char *fileNameLineas = "../../inputs/lineas";
    char *fileNameCargas = "../../inputs/cargas";
    char *fileNameGen = "../../inputs/gen";
    double *Vn,*An,t;
    structData *data;
    data = (structData*)malloc(sizeof(structData));
    res = loadDataFromFile(fileNameLineas,fileNameCargas,fileNameGen, data);
    data->maxIter = 100;
    data->numN = maxLineas(data,widthLineas,heightLineas);
    data->numL = heightLineas;
    data->numG = heightGen;
    data->numC = heightCargas;
    Vn = (double*)malloc(data->numN*sizeof(double));
    An = (double*)malloc(data->numN*sizeof(double));
    double *ybusReal = (double*) malloc(data->numN*data->numN*sizeof(double));
    double *ybusImag = (double*) malloc(data->numN*data->numN*sizeof(double));
    ones(data->numN,Vn);
    zeros(data->numN,An);
    calcularYbus(data,ybusReal,ybusImag);
    NumP = (int) data->numN - 1;
    int *NNP = (int *)malloc((data->numN-1)*sizeof(int));
    genVector(NNP, 2,data->numN);
    int *vector1 = (int*)malloc(data->numN*sizeof(int));
    genVector(vector1, 1, data->numN);
    int *NNQ = (int *) malloc(data->numN*sizeof(int));
    int NumQ = setdiff(vector1, data->gen, data->numN, data->numG, NNQ);
    double *Pref = (double*)malloc(data->numN*sizeof(double));
    double *Qref = (double*)malloc(data->numN*sizeof(double));
    zeros(data->numN,Pref);
    zeros(data->numN,Qref);
    int k;
    int N1;

    for (k = 0; k < data->numG; k++) {
        N1 = (int) data->gen[k*widthGen+0] - 1;
        Pref[N1] = Pref[N1] + data->gen[k*widthGen+1];
        Vn[N1] = data->gen[k*widthGen+2];
    }

    for (k = 0 ; k < data->numC; k++) {
        N1 = (int)data->cargas[k*widthCargas] - 1;
        Pref[N1] = Pref[N1] - data->cargas[k*widthCargas+1];
        Qref[N1] = Qref[N1] - data->cargas[k*widthCargas+2];
    }


    double *dP = (double*)malloc(NumP*sizeof(double));
    double *dQ = (double*)malloc(NumQ*sizeof(double));

    zeros(NumP,dP);
    zeros(NumQ,dQ);

    double Error = 100.0;
    int iter = 0;
    int lda = NumP+NumQ,kk;
    int NumPQ = NumP+NumQ, nrhs = 1;
    int *ipiv,ldb = NumQ+NumP,info;

    double *Jpp, *Jpq, *Jqp, *Jqq, *Pn, *Qn, *JacR, *dPdQ, *JacRt,*dX, *Ism;
    double *d_dX, *d_JacRt, *d_work;
    int *devIpiv;

    Jpp = (double*)malloc(data->numN*data->numN*sizeof(double));
    Jpq = (double*)malloc(data->numN*data->numN*sizeof(double));
    Jqp = (double*)malloc(data->numN*data->numN*sizeof(double));
    Jqq = (double*)malloc(data->numN*data->numN*sizeof(double));
    Pn = (double*)malloc(data->numN*sizeof(double));
    Qn = (double*)malloc(data->numN*sizeof(double));
    JacR = (double*)malloc((NumPQ)*(NumPQ)*sizeof(double));
    dPdQ = (double*)malloc((NumPQ)*sizeof(double));
    dX = (double*)malloc((NumPQ)*sizeof(double));
    ipiv = (int*)malloc((NumP+NumQ)*sizeof(int));
    JacRt = (double*)malloc((NumPQ)*(NumPQ)*sizeof(double));
    Ism = (double*)malloc(data->numL*sizeof(double));

    hipError_t error = hipSuccess;
    int lwork =0, *devInfo;
    gpuErrchk(hipMalloc(&devInfo, sizeof(int)));

    // ---- cuSolver initialization ---- //
    hipsolverStatus_t solvStatus = HIPSOLVER_STATUS_SUCCESS;
    hipsolverHandle_t handle;
    solvStatus = hipsolverDnCreate(&handle);
    ///////////////////////////////////////

    hipblasOperation_t trans = HIPBLAS_OP_N;


    gpuErrchk(hipMalloc((void**)&d_JacRt,sizeof(double)*NumPQ*NumPQ));
    gpuErrchk(hipMalloc((void**)&d_dX,sizeof(double)*NumPQ));
    gpuErrchk(hipMalloc((void**)&devIpiv,sizeof(int)*NumPQ));

    cusolveSafeCall(hipsolverDnDgetrf_bufferSize(handle,NumPQ,NumPQ,d_JacRt,lda,&lwork));

    gpuErrchk(hipMalloc((void**)&d_work, sizeof(double)*lwork));

    zeros(data->numL,Ism);

    while (Error>1e-8){
        calcularJacobiano(data,ybusReal,ybusImag,Vn,An,Jpp,Jpq,Jqp,Jqq,Pn,Qn);
        for (i = 0 ; i < NumP ; i++) {
            N1 = NNP[i] - 1;
            dP[i] = Pref[N1] - Pn[N1];
        }

        for (i = 0; i < NumQ; i++ ) {
            N1 = NNQ[i] - 1;
            dQ[i] = Qref[N1] - Qn[N1];
        }

        createJacR(NNP, NNQ, NumQ, NumP, (int)data->numN, Jpp, Jpq, Jqp, Jqq, JacR);
        transposeJacR(JacR,NumPQ,JacRt);
        createdPdQ(dP,dQ,NumP,NumQ,dPdQ);
        hipMemcpy(d_dX,dPdQ,sizeof(double)*NumPQ,hipMemcpyHostToDevice);
        hipMemcpy(d_JacRt,JacRt,sizeof(double)*NumPQ*NumPQ,hipMemcpyHostToDevice);

        cusolveSafeCall(hipsolverDnDgetrf(handle, NumPQ, NumPQ,d_JacRt,NumPQ,d_work,devIpiv,devInfo));
        gpuErrchk(hipDeviceSynchronize());
        cusolveSafeCall(hipsolverDnDgetrs(handle,trans,NumPQ,nrhs,d_JacRt,NumPQ,devIpiv,d_dX,NumPQ,devInfo));
        gpuErrchk(hipMemcpy(dX,d_dX,sizeof(double)*NumPQ,hipMemcpyDeviceToHost));

        for (k = 0; k < NumP; k++) {
            N1 = NNP[k] - 1;
            An[N1] = An[N1] + dX[k];
        }

        for (k = 0; k < NumQ; k++) {
            N1 = NNQ[k] - 1;
            kk = k + NumP;
            Vn[N1] = Vn[N1] + dX[kk];
        }

       Error = maxAbs(NumPQ,dPdQ);

        if (iter>data->maxIter) {
            printf("..... No converge despues de %d iteraciones\nError = %lf\n", data->maxIter, Error);
            break;
        }
        iter++;
    }

    calcCargLineas(data,An,Vn,Ism);
    printDataToFileVec("ismData",data->numL,Ism);
    printDataToFileVec("vnData",data->numN,Vn);
    printDataToFileVec("anData",data->numN,An);
    printDataToFileVec("pnData",data->numN,Pn);
    printDataToFileVec("qnData",data->numN,Qn);
    printDataToFileMat("ybusRealData",data->numN,ybusReal);
    printDataToFileMat("ybusImagData",data->numN,ybusImag);

    printDataToFileMat("jppData",data->numN,Jpp);
    printDataToFileMat("jpqData",data->numN,Jpq);
    printDataToFileMat("jqpData",data->numN,Jqp);
    printDataToFileMat("jqqData",data->numN,Jqq);


    hipsolverDnDestroy(handle);
    free(data);
    free(Vn);
    free(An);
    free(ybusReal);
    free(ybusImag);
    free(NNP);
    free(NNQ);
    free(vector1);
    free(dP);
    free(dQ);
    free(Pref);
    free(Qref);
    free(JacR);
    free(ipiv);
    free(JacRt);
    free(dX);
    free(Jpp);
    free(Jpq);
    free(Jqp);
    free(Jqq);
    free(Ism);
    hipFree(d_work);
    hipFree(d_JacRt);
    hipFree(devIpiv);
    hipFree(d_work);
    return res;
}
