#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include "utilities/utilities.h"
#include <hipsolver.h>
#include "utilities/cusolverUtilities.cuh"


int main(){
    int res,i,j;
    int widthCargas = 3;
    int widthGen = 3;
    int widthLineas = 6;
    int heightCargas = 83;
    int heightGen = 15;
    int heightLineas = 186, NumP;
    char *fileNameLineas = "../../inputs/lineas";
    char *fileNameCargas = "../../inputs/cargas";
    char *fileNameGen = "../../inputs/gen";
    double *Vn,*An,t;
    structData *data;
    data = (structData*)malloc(sizeof(structData));
    res = loadDataFromFile(fileNameLineas,fileNameCargas,fileNameGen, data);
    data->maxIter = 100;
    data->numN = maxLineas(data,widthLineas,heightLineas);
    data->numL = heightLineas;
    data->numG = heightGen;
    data->numC = heightCargas;
    Vn = (double*)malloc(data->numN*sizeof(double));
    An = (double*)malloc(data->numN*sizeof(double));
    double *ybusReal = (double*) malloc(data->numN*data->numN*sizeof(double));
    double *ybusImag = (double*) malloc(data->numN*data->numN*sizeof(double));
    ones(data->numN,Vn);
    zeros(data->numN,An);
    calcularYbus(data,ybusReal,ybusImag);
    NumP = (int) data->numN - 1;
    int *NNP = (int *)malloc((data->numN-1)*sizeof(int));
    genVector(NNP, 2,data->numN);
    int *vector1 = (int*)malloc(data->numN*sizeof(int));
    genVector(vector1, 1, data->numN);
    int *NNQ = (int *) malloc(data->numN*sizeof(int));
    int NumQ = setdiff(vector1, data->gen, data->numN, data->numG, NNQ);
    double *Pref = (double*)malloc(data->numN*sizeof(double));
    double *Qref = (double*)malloc(data->numN*sizeof(double));
    zeros(data->numN,Pref);
    zeros(data->numN,Qref);
    int k;
    int N1;

    for (k = 0; k < data->numG; k++) {
        N1 = (int) data->gen[k*widthGen+0] - 1;
        Pref[N1] = Pref[N1] + data->gen[k*widthGen+1];
        Vn[N1] = data->gen[k*widthGen+2];
    }

    for (k = 0 ; k < data->numC; k++) {
        N1 = (int)data->cargas[k*widthCargas] - 1;
        Pref[N1] = Pref[N1] - data->cargas[k*widthCargas+1];
        Qref[N1] = Qref[N1] - data->cargas[k*widthCargas+2];
    }


    double *dP = (double*)malloc(NumP*sizeof(double));
    double *dQ = (double*)malloc(NumQ*sizeof(double));

    zeros(NumP,dP);
    zeros(NumQ,dQ);

    double Error = 100.0;
    int iter = 0;
    int lda = NumP+NumQ,kk;
    int NumPQ = NumP+NumQ, nrhs = 1;
    int *ipiv,ldb = NumQ+NumP,info;

    double *Jpp, *Jpq, *Jqp, *Jqq, *Pn, *Qn, *JacR, *dPdQ, *JacRt,*dX, *Ism;
    double *d_Jpp, *d_Jpq, *d_Jqp, *d_Jqq, *d_Pn, *d_Qn, *d_ybusReal, *d_ybusImag, *d_Vn, *d_An;
    double *d_dX, *d_JacRt, *d_work;
    int *devIpiv;

    Jpp = (double*)malloc(data->numN*data->numN*sizeof(double));
    Jpq = (double*)malloc(data->numN*data->numN*sizeof(double));
    Jqp = (double*)malloc(data->numN*data->numN*sizeof(double));
    Jqq = (double*)malloc(data->numN*data->numN*sizeof(double));
    Pn = (double*)malloc(data->numN*sizeof(double));
    Qn = (double*)malloc(data->numN*sizeof(double));
    JacR = (double*)malloc((NumPQ)*(NumPQ)*sizeof(double));
    dPdQ = (double*)malloc((NumPQ)*sizeof(double));
    dX = (double*)malloc((NumPQ)*sizeof(double));
    ipiv = (int*)malloc((NumP+NumQ)*sizeof(int));
    JacRt = (double*)malloc((NumPQ)*(NumPQ)*sizeof(double));
    Ism = (double*)malloc(data->numL*sizeof(double));

    hipError_t error = hipSuccess;
    int lwork =0, *devInfo;
    gpuErrchk(hipMalloc(&devInfo, sizeof(int)));


    gpuErrchk(hipMalloc(&d_ybusImag,data->numN*data->numN*sizeof(double)));
    gpuErrchk(hipMalloc(&d_ybusReal,data->numN*data->numN*sizeof(double)));
    gpuErrchk(hipMalloc(&d_Qn,data->numN*sizeof(double)));
    gpuErrchk(hipMalloc(&d_Pn,data->numN*sizeof(double)));
    gpuErrchk(hipMalloc(&d_Jpp,data->numN*data->numN*sizeof(double)));
    gpuErrchk(hipMalloc(&d_Jqp,data->numN*data->numN*sizeof(double)));
    gpuErrchk(hipMalloc(&d_Jpq,data->numN*data->numN*sizeof(double)));
    gpuErrchk(hipMalloc(&d_Jqq,data->numN*data->numN*sizeof(double)));
    gpuErrchk(hipMalloc(&d_Vn,data->numN*sizeof(double)));
    gpuErrchk(hipMalloc(&d_An,data->numN*sizeof(double)));

    // ---- Copy ybusData to GPU ----//
    gpuErrchk(hipMemcpy(d_ybusReal,ybusReal,sizeof(double)*data->numN*data->numN,hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_ybusImag,ybusImag,sizeof(double)*data->numN*data->numN,hipMemcpyHostToDevice));

    // ---- cuSolver initialization ---- //
    hipsolverStatus_t solvStatus = HIPSOLVER_STATUS_SUCCESS;
    hipsolverHandle_t handle;
    solvStatus = hipsolverDnCreate(&handle);
    ///////////////////////////////////////

    hipblasOperation_t trans = HIPBLAS_OP_N;


    gpuErrchk(hipMalloc((void**)&d_JacRt,sizeof(double)*NumPQ*NumPQ));
    gpuErrchk(hipMalloc((void**)&d_dX,sizeof(double)*NumPQ));
    gpuErrchk(hipMalloc((void**)&devIpiv,sizeof(int)*NumPQ));

    cusolveSafeCall(hipsolverDnDgetrf_bufferSize(handle,NumPQ,NumPQ,d_JacRt,lda,&lwork));

    gpuErrchk(hipMalloc((void**)&d_work, sizeof(double)*lwork));

    zeros(data->numL,Ism);

    int blockSize = 32;
    dim3 dimBlock(blockSize,1,1);
    dim3 dimGrid(ceil(data->numN/float(blockSize)),1,1);

    dim3 dimBlock2(blockSize,blockSize,1);
    dim3 dimGrid2(ceil(data->numN/float(blockSize)),ceil(data->numN/float(blockSize)),1);

    t = 0;
   while (Error>1e-8){
         // ---- Copy Vn, An data to GPU ----//

       zeros(data->numN*data->numN,Jpp);
       zeros(data->numN*data->numN,Jpq);
       zeros(data->numN*data->numN,Jqp);
       zeros(data->numN*data->numN,Jqq);

        gpuErrchk(hipMemcpy(d_Vn,Vn,sizeof(double)*data->numN,hipMemcpyHostToDevice));
        gpuErrchk(hipMemcpy(d_An,An,sizeof(double)*data->numN,hipMemcpyHostToDevice));

        gpuErrchk(hipMemcpy(d_Jpp,Vn,sizeof(double)*data->numN*data->numN,hipMemcpyHostToDevice));
        gpuErrchk(hipMemcpy(d_Jqp,Vn,sizeof(double)*data->numN*data->numN,hipMemcpyHostToDevice));
        gpuErrchk(hipMemcpy(d_Jpq,Vn,sizeof(double)*data->numN*data->numN,hipMemcpyHostToDevice));
        gpuErrchk(hipMemcpy(d_Jqq,Vn,sizeof(double)*data->numN*data->numN,hipMemcpyHostToDevice));

       //calcularJacobiano(data,ybusReal,ybusImag,Vn,An,Jpp,Jpq,Jqp,Jqq,Pn,Qn);
        d_calcularJacobiano_1<<<dimGrid,dimBlock>>>(data->numN, d_ybusReal, d_ybusImag,d_Vn,d_An,d_Pn,d_Qn);
        hipDeviceSynchronize();
        d_calcularJacobiano_2<<<dimGrid2,dimBlock2>>>(data->numN, d_ybusReal, d_ybusImag, d_Vn,d_An, d_Pn,d_Qn, d_Jpp, d_Jpq, d_Jqp, d_Jqq);
        hipDeviceSynchronize();

        gpuErrchk(hipMemcpy(Pn,d_Pn,sizeof(double)*data->numN,hipMemcpyDeviceToHost));
        gpuErrchk(hipMemcpy(Qn,d_Qn,sizeof(double)*data->numN,hipMemcpyDeviceToHost));
        gpuErrchk(hipMemcpy(Jpp,d_Jpp,sizeof(double)*data->numN*data->numN,hipMemcpyDeviceToHost));
        gpuErrchk(hipMemcpy(Jpq,d_Jpq,sizeof(double)*data->numN*data->numN,hipMemcpyDeviceToHost));
        gpuErrchk(hipMemcpy(Jqp,d_Jqp,sizeof(double)*data->numN*data->numN,hipMemcpyDeviceToHost));
        gpuErrchk(hipMemcpy(Jqq,d_Jqq,sizeof(double)*data->numN*data->numN,hipMemcpyDeviceToHost));
        gpuErrchk(hipMemcpy(An,d_An,sizeof(double)*data->numN,hipMemcpyDeviceToHost));
        gpuErrchk(hipMemcpy(Vn,d_Vn,sizeof(double)*data->numN,hipMemcpyDeviceToHost));

        if(t==0)
        {
            printDataToFileVec("pnData",data->numN,Pn);
            printDataToFileMat("jppData",data->numN,Jpp);
            printDataToFileMat("jpqData",data->numN,Jpq);
            printDataToFileMat("jqpData",data->numN,Jqp);
            printDataToFileMat("jqqData",data->numN,Jqq);

            printDataToFileVec("qnData",data->numN,Qn);
            t=1;
        }
        for (i = 0 ; i < NumP ; i++) {
            N1 = NNP[i] - 1;
            dP[i] = Pref[N1] - Pn[N1];
        }

        for (i = 0; i < NumQ; i++ ) {
            N1 = NNQ[i] - 1;
            dQ[i] = Qref[N1] - Qn[N1];
        }

        createJacR(NNP, NNQ, NumQ, NumP, (int)data->numN, Jpp, Jpq, Jqp, Jqq, JacR);
        transposeJacR(JacR,NumPQ,JacRt);
        createdPdQ(dP,dQ,NumP,NumQ,dPdQ);
        hipMemcpy(d_dX,dPdQ,sizeof(double)*NumPQ,hipMemcpyHostToDevice);
        hipMemcpy(d_JacRt,JacRt,sizeof(double)*NumPQ*NumPQ,hipMemcpyHostToDevice);

        cusolveSafeCall(hipsolverDnDgetrf(handle, NumPQ, NumPQ,d_JacRt,NumPQ,d_work,devIpiv,devInfo));
        gpuErrchk(hipDeviceSynchronize());
        cusolveSafeCall(hipsolverDnDgetrs(handle,trans,NumPQ,nrhs,d_JacRt,NumPQ,devIpiv,d_dX,NumPQ,devInfo));
        gpuErrchk(hipMemcpy(dX,d_dX,sizeof(double)*NumPQ,hipMemcpyDeviceToHost));

        for (k = 0; k < NumP; k++) {
            N1 = NNP[k] - 1;
            An[N1] = An[N1] + dX[k];
        }

        for (k = 0; k < NumQ; k++) {
            N1 = NNQ[k] - 1;
            kk = k + NumP;
            Vn[N1] = Vn[N1] + dX[kk];
        }

       Error = maxAbs(NumPQ,dPdQ);

        if (iter>data->maxIter) {
            printf("..... No converge despues de %d iteraciones\nError = %lf\n", data->maxIter, Error);
            break;
        }
        iter++;
    }

    calcCargLineas(data,An,Vn,Ism);
    printDataToFileVec("ismData",data->numL,Ism);
    printDataToFileVec("vnData",data->numN,Vn);
    printDataToFileVec("anData",data->numN,An);
   // printDataToFileVec("pnData",data->numN,Pn);
   // printDataToFileVec("qnData",data->numN,Qn);
    printDataToFileMat("ybusRealData",data->numN,ybusReal);
    printDataToFileMat("ybusImagData",data->numN,ybusImag);

/*    printDataToFileMat("jppData",data->numN,Jpp);
    printDataToFileMat("jpqData",data->numN,Jpq);
    printDataToFileMat("jqpData",data->numN,Jqp);
    printDataToFileMat("jqqData",data->numN,Jqq);*/


    hipsolverDnDestroy(handle);
    free(data);
    free(Vn);
    free(An);
    free(ybusReal);
    free(ybusImag);
    free(NNP);
    free(NNQ);
    free(vector1);
    free(dP);
    free(dQ);
    free(Pref);
    free(Qref);
    free(JacR);
    free(ipiv);
    free(JacRt);
    free(dX);
    free(Jpp);
    free(Jpq);
    free(Jqp);
    free(Jqq);
    free(Ism);
    hipFree(d_ybusReal);
    hipFree(d_ybusImag);
    hipFree(d_work);
    hipFree(d_JacRt);
    hipFree(devIpiv);
    hipFree(d_work);
    hipFree(d_Jpp);
    hipFree(d_Jqq);
    hipFree(d_Jpq);
    hipFree(d_Jqp);
    hipFree(d_Vn);
    hipFree(d_An);
    return res;
}
