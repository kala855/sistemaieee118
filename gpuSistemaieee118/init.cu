#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include "utilities/utilities.h"
#include <hipsolver.h>
#include "utilities/cusolverUtilities.cuh"

/********************/
/* CUDA ERROR CHECK */
/********************/
// --- Credit to http://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) { exit(code); }
    }
}

void gpuErrchk(hipError_t ans) { gpuAssert((ans), __FILE__, __LINE__); }



int main(){
    int res,i,j;
    int widthCargas = 3;
    int widthGen = 3;
    int widthLineas = 6;
    int heightCargas = 83*10;
    int heightGen = 15*10;
    int heightLineas = 1896/*186*/, NumP;
    char *fileNameLineas = "../../inputs/lineasBig";
    char *fileNameCargas = "../../inputs/cargasBig";
    char *fileNameGen = "../../inputs/genBig";
    double *Vn,*An,t;
    structData *data;
    data = (structData*)malloc(sizeof(structData));
    res = loadDataFromFile(fileNameLineas,fileNameCargas,fileNameGen, data);
    data->maxIter = 100;
    data->numN = maxLineas(data,widthLineas,heightLineas);
    data->numL = heightLineas;
    data->numG = heightGen;
    data->numC = heightCargas;
    Vn = (double*)malloc(data->numN*sizeof(double));
    An = (double*)malloc(data->numN*sizeof(double));
    double *ybusReal = (double*) malloc(data->numN*data->numN*sizeof(double));
    double *ybusImag = (double*) malloc(data->numN*data->numN*sizeof(double));
    ones(data->numN,Vn);
    zeros(data->numN,An);
    calcularYbus(data,ybusReal,ybusImag);
    NumP = (int) data->numN - 1;
    int *NNP = (int *)malloc((data->numN-1)*sizeof(int));
    genVector(NNP, 2,data->numN);
    int *vector1 = (int*)malloc(data->numN*sizeof(int));
    genVector(vector1, 1, data->numN);
    int *NNQ = (int *) malloc(data->numN*sizeof(int));
    int NumQ = setdiff(vector1, data->gen, data->numN, data->numG, NNQ);
    double *Pref = (double*)malloc(data->numN*sizeof(double));
    double *Qref = (double*)malloc(data->numN*sizeof(double));
    zeros(data->numN,Pref);
    zeros(data->numN,Qref);
    int k;
    int N1;

    for (k = 0; k < data->numG; k++) {
        N1 = (int) data->gen[k*widthGen+0] - 1;
        Pref[N1] = Pref[N1] + data->gen[k*widthGen+1];
        Vn[N1] = data->gen[k*widthGen+2];
    }

    for (k = 0 ; k < data->numC; k++) {
        N1 = (int)data->cargas[k*widthCargas] - 1;
        Pref[N1] = Pref[N1] - data->cargas[k*widthCargas+1];
        Qref[N1] = Qref[N1] - data->cargas[k*widthCargas+2];
    }


    double *dP = (double*)malloc(NumP*sizeof(double));
    double *dQ = (double*)malloc(NumQ*sizeof(double));

    zeros(NumP,dP);
    zeros(NumQ,dQ);

    double Error = 100.0;
    int iter = 0;
    int lda = NumP+NumQ,kk;
    int NumPQ = NumP+NumQ, nrhs = 1;
    int *ipiv,ldb = NumQ+NumP,info;

    double *Jpp, *Jpq, *Jqp, *Jqq, *Pn, *Qn, *JacR, *dPdQ, *JacRt,*dX, *Ism;
    double *d_Jpp, *d_Jpq, *d_Jqp, *d_Jqq, *d_Pn, *d_Qn, *d_ybusReal, *d_ybusImag, *d_Vn, *d_An;
    double *d_dX, *d_JacRt, *d_work, *d_dP, *d_dQ, *d_Pref, *d_Qref, *d_JacR, *d_dPdQ;
    int *devIpiv, *d_NNP, *d_NNQ;

    Jpp = (double*)malloc(data->numN*data->numN*sizeof(double));
    Jpq = (double*)malloc(data->numN*data->numN*sizeof(double));
    Jqp = (double*)malloc(data->numN*data->numN*sizeof(double));
    Jqq = (double*)malloc(data->numN*data->numN*sizeof(double));
    Pn = (double*)malloc(data->numN*sizeof(double));
    Qn = (double*)malloc(data->numN*sizeof(double));
    JacR = (double*)malloc((NumPQ)*(NumPQ)*sizeof(double));
    dPdQ = (double*)malloc((NumPQ)*sizeof(double));
    dX = (double*)malloc((NumPQ)*sizeof(double));
    ipiv = (int*)malloc((NumP+NumQ)*sizeof(int));
    JacRt = (double*)malloc((NumPQ)*(NumPQ)*sizeof(double));
    Ism = (double*)malloc(data->numL*sizeof(double));

    hipError_t error = hipSuccess;
    int lwork =0, *devInfo;
    gpuErrchk(hipMalloc(&devInfo, sizeof(int)));


    gpuErrchk(hipMalloc(&d_ybusImag,data->numN*data->numN*sizeof(double)));
    gpuErrchk(hipMalloc(&d_ybusReal,data->numN*data->numN*sizeof(double)));
    gpuErrchk(hipMalloc(&d_Qn,data->numN*sizeof(double)));
    gpuErrchk(hipMalloc(&d_Pn,data->numN*sizeof(double)));
    gpuErrchk(hipMalloc(&d_Jpp,data->numN*data->numN*sizeof(double)));
    gpuErrchk(hipMalloc(&d_Jqp,data->numN*data->numN*sizeof(double)));
    gpuErrchk(hipMalloc(&d_Jpq,data->numN*data->numN*sizeof(double)));
    gpuErrchk(hipMalloc(&d_Jqq,data->numN*data->numN*sizeof(double)));
    gpuErrchk(hipMalloc(&d_Vn,data->numN*sizeof(double)));
    gpuErrchk(hipMalloc(&d_An,data->numN*sizeof(double)));
    gpuErrchk(hipMalloc(&d_dP,NumP*sizeof(double)));
    gpuErrchk(hipMalloc(&d_dQ,NumQ*sizeof(double)));
    gpuErrchk(hipMalloc(&d_Pref,(data->numN)*sizeof(double)));
    gpuErrchk(hipMalloc(&d_Qref,(data->numN)*sizeof(double)));
    gpuErrchk(hipMalloc(&d_NNP,((data->numN)-1)*sizeof(int)));
    gpuErrchk(hipMalloc(&d_NNQ,(data->numN)*sizeof(int)));
    gpuErrchk(hipMalloc(&d_JacR,NumPQ*NumPQ*sizeof(double)));
    gpuErrchk(hipMalloc(&d_dPdQ,NumPQ*sizeof(double)));


    // ---- Copy ybusData to GPU ----//
    gpuErrchk(hipMemcpy(d_ybusReal,ybusReal,sizeof(double)*data->numN*data->numN\
                ,hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_ybusImag,ybusImag,sizeof(double)*data->numN*data->numN\
                ,hipMemcpyHostToDevice));
    /*---- Copy Pref and Qref to device ----*/
    gpuErrchk(hipMemcpy(d_Pref,Pref,sizeof(double)*data->numN,hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_Qref,Qref,sizeof(double)*data->numN,hipMemcpyHostToDevice));

    /*---- Copy Pref and Qref to device ----*/
    gpuErrchk(hipMemcpy(d_NNP,NNP,sizeof(int)*((data->numN)-1),hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_NNQ,NNQ,sizeof(int)*data->numN,hipMemcpyHostToDevice));
    /*---- Copy An y Vn al device -----*/
    gpuErrchk(hipMemcpy(d_Vn,Vn,sizeof(double)*data->numN,hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_An,An,sizeof(double)*data->numN,hipMemcpyHostToDevice));


    // ---- cuSolver initialization ---- //
    hipsolverStatus_t solvStatus = HIPSOLVER_STATUS_SUCCESS;
    hipsolverHandle_t handle;
    solvStatus = hipsolverDnCreate(&handle);
    ///////////////////////////////////////

    hipblasOperation_t trans = HIPBLAS_OP_N;

    gpuErrchk(hipMalloc((void**)&d_JacRt,sizeof(double)*NumPQ*NumPQ));
    gpuErrchk(hipMalloc((void**)&d_dX,sizeof(double)*NumPQ));
    gpuErrchk(hipMalloc((void**)&devIpiv,sizeof(int)*NumPQ));

    cusolveSafeCall(hipsolverDnDgetrf_bufferSize(handle,NumPQ,NumPQ,d_JacRt,lda,&lwork));

    gpuErrchk(hipMalloc((void**)&d_work, sizeof(double)*lwork));

    zeros(data->numL,Ism);

    int blockSize2D = 32;
    int blockSize = 1024;
    dim3 dimBlock(blockSize,1,1);
    dim3 dimGrid(ceil(data->numN/float(blockSize)),1,1);

    dim3 dimBlockXY(blockSize2D,blockSize2D,1);
    dim3 dimGridXY(ceil(data->numN/float(blockSize2D)),ceil(data->numN/float(blockSize2D)),1);

    dim3 dimBlock2(blockSize2D,blockSize2D,1);
    dim3 dimGrid2(ceil(data->numN/float(blockSize2D)),ceil(data->numN/float(blockSize2D)),1);

    dim3 dimGrid3(ceil((data->numN*data->numN)/float(blockSize)),1,1);
    dim3 dimGrid4(ceil(NumP/float(blockSize)),1,1);
    dim3 dimGrid5(ceil(NumQ/float(blockSize)),1,1);
    dim3 dimGrid6(ceil(NumP/float(blockSize2D)), ceil(NumP/float(blockSize2D)),1);
    dim3 dimGrid7(ceil(NumP/float(blockSize)),1,1);
    dim3 dimGrid8(ceil(NumPQ/float(blockSize2D)),ceil(NumPQ/float(blockSize2D)),1);
    dim3 dimGrid9(ceil(NumQ/float(blockSize)),1,1);
    dim3 dimGrid10(ceil(NumPQ/float(blockSize)),1,1);

    while (Error>1e-8){

        /*---- Initialize d_Jpp, d_Jpq, d_Jqp, d_Jqq, ----*/
        d_zeros<<<dimGrid3,dimBlock>>>(data->numN*data->numN,d_Jpp);
        d_zeros<<<dimGrid3,dimBlock>>>(data->numN*data->numN,d_Jpq);
        d_zeros<<<dimGrid3,dimBlock>>>(data->numN*data->numN,d_Jqp);
        d_zeros<<<dimGrid3,dimBlock>>>(data->numN*data->numN,d_Jqq);
        hipDeviceSynchronize();

        d_calcularJacobiano_1<<<dimGrid,dimBlock>>>(data->numN, d_ybusReal, d_ybusImag,d_Vn,\
                d_An,d_Pn,d_Qn);

        hipDeviceSynchronize();
        d_calcularJacobiano_2<<<dimGrid2,dimBlock2>>>(data->numN, d_ybusReal, d_ybusImag, d_Vn\
                ,d_An, d_Pn,d_Qn, d_Jpp, d_Jpq, d_Jqp, d_Jqq);
        hipDeviceSynchronize();

        dp_compute<<<dimGrid4,dimBlock>>>(NumP, d_NNP, d_Pref, d_Pn, d_dP);

        dq_compute<<<dimGrid5,dimBlock>>>(NumP, d_NNQ, d_Qref, d_Qn, d_dQ);

        d_createJacR_1<<<dimGrid6,dimBlock2>>>(d_NNP, NumQ, NumP, (int)(data->numN), d_Jpp, d_JacR);
        hipDeviceSynchronize();
        d_createJacR_2<<<dimGrid7,dimBlock>>>(d_NNP, d_NNQ, NumQ, NumP, (int)(data->numN), d_Jpq, \
                d_JacR);
        hipDeviceSynchronize();
        d_createJacR_3<<<dimGrid7,dimBlock>>>(d_NNP, d_NNQ, NumQ, NumP,(int)(data->numN), d_Jqp, \
                d_JacR);
        hipDeviceSynchronize();
        d_createJacR_4<<<dimGrid5,dimBlock>>>(d_NNQ, NumQ, NumP,(int)(data->numN),\
                d_Jqq, d_JacR);
        hipDeviceSynchronize();


        d_transposeJacr<<<dimGrid8,dimBlock2>>>(d_JacR, NumPQ, d_JacRt);
        hipDeviceSynchronize();

        d_filldPdQ1<<<dimGrid7,dimBlock>>>(d_dP,NumP,d_dPdQ);
        hipDeviceSynchronize();
        d_filldPdQ<<<dimGrid9,dimBlock>>>(d_dQ,NumQ,NumP,d_dPdQ);
        hipDeviceSynchronize();

        d_fill_d_dx<<<dimGrid10,dimBlock>>>(d_dPdQ,NumPQ,d_dX);
        hipDeviceSynchronize();


        cusolveSafeCall(hipsolverDnDgetrf(handle, NumPQ, NumPQ,d_JacRt,NumPQ,d_work,devIpiv,devInfo));
        gpuErrchk(hipDeviceSynchronize());
        cusolveSafeCall(hipsolverDnDgetrs(handle,trans,NumPQ,nrhs,d_JacRt,NumPQ,devIpiv,d_dX,NumPQ,\
                    devInfo));
        hipDeviceSynchronize();


        d_calc_An<<<dimGrid7,dimBlock>>>(d_dX, d_NNP, NumP, d_An);
        hipDeviceSynchronize();

        d_calc_Vn<<<dimGrid9,dimBlock>>>(d_dX, d_NNQ, NumP, NumQ, d_Vn);
        hipDeviceSynchronize();

        gpuErrchk(hipMemcpy(dPdQ, d_dPdQ,sizeof(double)*NumPQ,hipMemcpyDeviceToHost));
        Error = maxAbs(NumPQ,dPdQ);

        if (iter>data->maxIter) {
            printf("..... No converge despues de %d iteraciones\nError = %lf\n", data->maxIter, Error);
            break;
        }
        iter++;
    }

    gpuErrchk(hipMemcpy(Pn,d_Pn,sizeof(double)*data->numN,hipMemcpyDeviceToHost));
    printDataToFileVec("pnData",data->numN,Pn);

    gpuErrchk(hipMemcpy(An,d_An,sizeof(double)*data->numN,hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(Vn,d_Vn,sizeof(double)*data->numN,hipMemcpyDeviceToHost));
    calcCargLineas(data,An,Vn,Ism);
    printDataToFileVec("ismData",data->numL,Ism);
    printDataToFileVec("vnData",data->numN,Vn);
    printDataToFileVec("anData",data->numN,An);

    hipsolverDnDestroy(handle);
    free(data);
    free(Vn);
    free(An);
    free(ybusReal);
    free(ybusImag);
    free(NNP);
    free(NNQ);
    free(vector1);
    free(dP);
    free(dQ);
    free(Pref);
    free(Qref);
    free(JacR);
    free(ipiv);
    free(JacRt);
    free(dX);
    free(Jpp);
    free(Jpq);
    free(Jqp);
    free(Jqq);
    free(Ism);
    hipFree(d_ybusReal);
    hipFree(d_ybusImag);
    hipFree(d_work);
    hipFree(d_JacRt);
    hipFree(devIpiv);
    hipFree(d_work);
    hipFree(d_Jpp);
    hipFree(d_Jqq);
    hipFree(d_Jpq);
    hipFree(d_Jqp);
    hipFree(d_Vn);
    hipFree(d_An);
    hipFree(d_Pref);
    hipFree(d_Qref);
    hipFree(d_dP);
    hipFree(d_dQ);
    hipFree(d_JacR);
    hipFree(d_JacRt);
    hipFree(d_dPdQ);
    return res;
}
